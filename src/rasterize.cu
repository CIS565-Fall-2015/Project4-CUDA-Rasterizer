#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"
#include <glm/gtc/matrix_transform.hpp>


struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
	glm::vec3 pos;
    // TODO
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;
static glm::mat4 matrix;

//Things added
static VertexOut *dev_outVertex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

__global__
void kernVertexShader(int numVertices, int w, int h, VertexIn * inVertex, VertexOut *outVertex, glm::mat4 matrix)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numVertices)
	{
		glm::vec4 outPoint = glm::vec4(inVertex[index].pos.x, inVertex[index].pos.y, inVertex[index].pos.z, 1.0f);

		outPoint = matrix * outPoint;

//		printf("OutPoint : %f %f %f %f\n", outPoint.x, outPoint.y, outPoint.z, outPoint.w);

		if(outPoint.w != 0)
			outVertex[index].pos = glm::vec3(outPoint / outPoint.w);
//		printf ("InVertex : %f %f \nOutVertex : %f %f \n\n", inVertex[index].pos.x, inVertex[index].pos.y, outVertex[index].pos.x, outVertex[index].pos.y);
	}
}

__global__
void kernPrimitiveAssembly(int numTriangles, VertexOut *outVertex, Triangle *triangles, int* indices)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		int k_3 = 3 * index;
		triangles[index].v[0] = outVertex[indices[k_3]];
		triangles[index].v[1] = outVertex[indices[k_3+1]];
		triangles[index].v[2] = outVertex[indices[k_3+2]];

//		printf ("Triangle : %d\n", index);
//		printf ("Vertex 1 : %f %f\n", triangles[index].v[0].pos.x, triangles[index].v[0].pos.y);
//		printf ("Vertex 2 : %f %f\n", triangles[index].v[1].pos.x, triangles[index].v[1].pos.y);
//		printf ("Vertex 3 : %f %f\n", triangles[index].v[2].pos.x, triangles[index].v[2].pos.y);
	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    hipFree(dev_outVertex);
    hipMalloc((void**)&dev_outVertex, vertCount * sizeof(VertexOut));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
bool run = true;

void createCamera()
{
	//Camera stuff
	glm::vec3 camEye, camCenter, camUp;
	camEye = glm::vec3(0,0,-2);
	camCenter = glm::vec3(0,0,0);
	camUp = glm::vec3(0,1,0);

	glm::mat4 view = glm::lookAt(camEye, camCenter, camUp);
//	glm::mat4 projection = glm::frustum<float>(-1, 1, -1, 1, -1, 1);
	glm::mat4 projection = glm::perspective<float>(45.0f, float(width)/ float(height), 0.1f, 100.0f);
	glm::mat4 model = glm::mat4();
	glm::mat4 temp;

//	std::cout<<"View : "<<std::endl;
//	utilityCore::printMat4(view);
//	std::cout<<std::endl<<"Projection : "<<std::endl;
//	utilityCore::printMat4(projection);
//	std::cout<<std::endl<<"Model : "<<std::endl;
//	utilityCore::printMat4(model);
//	std::cout<<std::endl;

	matrix = projection * view * model;
}

void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);


    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

    int numTriangles = vertCount/3;

    if(run)
    {
    	createCamera();
//    	run = false;
    }

    //Todo change the number of threads based on input size.
    if(run)
    {
    	kernVertexShader<<<1, vertCount>>>(vertCount, width, height, dev_bufVertex, dev_outVertex, matrix);
//    	run = false;
    }

    if(run)
    {
    	kernPrimitiveAssembly<<<1, numTriangles>>>(numTriangles, dev_outVertex, dev_primitives, dev_bufIdx);
    	run = false;
    }

    if(run)
    {
    	//kernRasterize
    }

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    hipFree(dev_outVertex);
    dev_outVertex = NULL;
//
//    hipFree(dev_view);
//    dev_view = NULL;
//
//    hipFree(dev_model);
//    dev_model = NULL;
//
//    hipFree(dev_projection);
//    dev_projection = NULL;

    checkCUDAError("rasterizeFree");
}
