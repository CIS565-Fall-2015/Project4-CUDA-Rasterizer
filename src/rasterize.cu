#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
    // TODO: Mirrors VertexIn?
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
	// will probably need to add more here? Will require updating my clear method
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL; //TODO Shouldn't this really be changed to indicate that it is in?
static VertexOut *dev_bufVertexOut = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/**
 * Clears the depth buffer with a black color.
 */
__global__
void clearDepthBuffer(int w, int h, Fragment *depthbuffer) {
	// TODO: Block stuff wrong?
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		depthbuffer[index].color = glm::vec3(0.0f);
	}
}

/**
 * Apply vertex transformations and transfer to vertex out buffer
 */
__global__
void vertexShading(int w, int h, VertexIn *vertexBufferIn, VertexOut *vertexBufferOut) {
	// TODO: Block stuff wrong?
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	// in this iteration just copy shit over. Later iterations will do transforms
	if (x < w && y < h) {
		vertexBufferOut[index].col = vertexBufferIn[index].col;
		vertexBufferOut[index].nor = vertexBufferIn[index].nor;
		vertexBufferOut[index].pos = vertexBufferIn[index].pos;
	}
}

/**
 * Assemble primitives from vertex out buffer data.
 */
__global__
void assemblePrimitives(int w, int h, VertexOut *vertexBufferOut, Triangle *primitives) {
	// TODO: Block stuff is WAY wrong here. 
	// Currently only supports triangles
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		Triangle primitive = primitives[index / 3];
		for (int i = 0; i < 3; i++) {
			primitive.v[i] = vertexBufferOut[3 * index + i]; // three times as many verts as triangles, gotta offset by index
			// You're getting the 3 verts that make up a triangle
		}
	}
}

__global__
void scanlineRaserization(int w, int h, Triangle *primitives, Fragment *depthbuffer) {
	// this is going to be the most incomplete one.
	// how large is hte depthbuffer vs the number of primitives? how do I insert them?
	// which buffer are they going to right now?
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	hipFree(dev_bufVertexOut);
	hipMalloc(&dev_bufVertexOut, vertCount * sizeof(VertexOut));
	hipMemset(dev_bufVertexOut, 0, vertCount * sizeof(VertexIn));

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)
	

	//so each stage of the pipeline is just anotehr kernel function?
	// Or should they be combined somewhat into one or two kerns?

	// first clear the depth buffer with some default value (0?)
	// TODO: Block stuff wrong?
	clearDepthBuffer<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);

	// Vertex shading
	// TODO: Block stuff wrong?
	vertexShading<<<blockCount2d, blockSize2d>>>(width, height, dev_bufVertex, dev_bufVertexOut);

	// Primitive Assembly
	// TODO: Block stuff wrong?
	assemblePrimitives<<<blockCount2d, blockSize2d>>>(width, height, dev_bufVertexOut, dev_primitives);

	// rasterization

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
