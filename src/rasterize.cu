#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <iostream>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
    // TODO
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static int *dev_depth=NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static VertexOut *dev_vertexOut=NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

__global__ void vertexShadingTest(VertexIn *vs_input,VertexOut *vs_output,int N){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<N){
		vs_output[index].col=vs_input[index].col;
		vs_output[index].nor=vs_input[index].nor;
		vs_output[index].pos=vs_input[index].pos;
	}
}

__global__ void vertexShading(VertexIn *vs_input,VertexOut *vs_output,
							  glm::vec3 cameraUp,glm::vec3 cameraFront,float fovy,float cameraDis,float rotation,int N){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<N){
		vs_output[index].col=vs_input[index].col;
		vs_output[index].nor=vs_input[index].nor;
		glm::mat4 m;
		//m=glm::translate(m,glm::vec3(0,0,1));
        //m=m*glm::rotate(m, glm::radians(rotation), glm::vec3(0.0f, 1.0f, 0.0f));
		m=m*glm::perspective(glm::radians(fovy),1.0f,0.1f,100.0f);
		m=m*glm::lookAt(-cameraFront,glm::vec3(0,0,0),cameraUp);
		//m=m*glm::rotate(m, glm::radians(rotation), glm::vec3(0.0f, 1.0f, 0.0f));
		vs_output[index].pos=multiplyMV(m,glm::vec4(vs_input[index].pos,1));
		vs_output[index].pos/=cameraDis;
		if(vs_output[index].pos.x<-1) vs_output[index].pos.x=-1;
		if(vs_output[index].pos.x>1) vs_output[index].pos.x=1;
		if(vs_output[index].pos.y<-1) vs_output[index].pos.y=-1;
		if(vs_output[index].pos.y>1) vs_output[index].pos.y=1;
		if(vs_output[index].pos.z<-1) vs_output[index].pos.z=-1;
		if(vs_output[index].pos.z>1) vs_output[index].pos.z=1;
	}
}

__global__ void primitiveAssemblyTest(VertexOut *vs_output,int *indices,Triangle *primitives,int N){
	int index=blockIdx.x*blockDim.x+threadIdx.x;
	if(index<N){
		primitives[index].v[0]=vs_output[indices[3*index]];
		primitives[index].v[1]=vs_output[indices[3*index+1]];
		primitives[index].v[2]=vs_output[indices[3*index+2]];
	}
}

__global__ void setColorToBlack(Fragment *fg_out,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		fg_out[index].color=glm::vec3(0,0,0);
	}
}

__global__ void setFlagZero(int *flag,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		flag[index]=0;
	}
}

__global__ void setDepthMax(int *depth,int N){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		depth[index]=1e12;
	}
}

__global__ void rasterizationTest(Triangle *primitives,Fragment *fg_out,int N,int Len){//no race condition considered, since only one triangle here.
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		float _len=2.0/Len;
		glm::vec3 tri[3];
		tri[0]=primitives[index].v[0].pos;
		tri[1]=primitives[index].v[1].pos;
		tri[2]=primitives[index].v[2].pos;
		AABB boundary=getAABBForTriangle(tri);
		for(int i=(1-boundary.max.y)/_len;i<(1-boundary.min.y)/_len;++i){
			for(int j=(boundary.min.x+1)/_len;j<(boundary.max.x+1)/_len;++j){
				glm::vec2 p(j*_len-1,1-i*_len);
				glm::vec3 tmp=calculateBarycentricCoordinate(tri,p);
				if(isBarycentricCoordInBounds(tmp)){
					fg_out[i*Len+j].color=glm::vec3(1,1,1);
				}
			}
		}
	}
}

__global__ void rasterization(Triangle *primitives,Fragment *fg_out,int *depth,glm::vec3 lightPos,glm::vec3 eyePos,int N,int Len){
	int index=blockDim.x*blockIdx.x+threadIdx.x;
	if(index<N){
		float _len=2.0/Len;
		glm::vec3 tri[3];
		tri[0]=primitives[index].v[0].pos;
		tri[1]=primitives[index].v[1].pos;
		tri[2]=primitives[index].v[2].pos;
		AABB boundary=getAABBForTriangle(tri);
		for(int i=(1-boundary.max.y)/_len;i<(1-boundary.min.y)/_len;++i){
			for(int j=(boundary.min.x+1)/_len;j<(boundary.max.x+1)/_len;++j){
				glm::vec2 p(j*_len-1,1-i*_len);
				glm::vec3 tmp=calculateBarycentricCoordinate(tri,p);
				if(isBarycentricCoordInBounds(tmp)){
					int currentDepth=(int)(1e6*getZAtCoordinate(tmp,tri));
					atomicMin(&depth[i*Len+j],currentDepth);
					if(currentDepth==depth[i*Len+j]){
						glm::vec3 n1=primitives[index].v[0].nor;
						glm::vec3 n2=primitives[index].v[1].nor;
						glm::vec3 n3=primitives[index].v[2].nor;
						glm::vec3 normal=n1*tmp.x+n2*tmp.y+n3*tmp.z;

						glm::vec3 pos=tri[0]*tmp.x+tri[1]*tmp.y+tri[2]*tmp.z;
						glm::vec3 dir=glm::normalize(lightPos-pos);
						glm::vec3 diffuse=glm::vec3(1,1,1)*(glm::dot(dir,normal));
						//diffuse

						glm::vec3 ref=dir-2.0f*normal*glm::dot(dir,normal);
						glm::vec3 eyeDir=glm::normalize(-eyePos-pos);
						glm::vec3 specular=glm::vec3(1,1,1)*max(0.0f,(float)pow(glm::dot(ref,eyeDir),20.0f));

						fg_out[i*Len+j].color=diffuse*0.7f+specular*0.3f;
						//fg_out[i*Len+j].color=normal;
						//fg_out[i*Len+j].color=glm::vec3(1,1,1);
					}
				}
			}
		}
	}
}

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
	hipMalloc(&dev_depth, width*height*sizeof(int));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);
	//std::cout<<vertCount<<std::endl;
    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
		//std::cout<<bufVertex[i].nor.x<<","<<bufVertex[i].nor.y<<","<<bufVertex[i].nor.z<<std::endl;
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	hipFree(dev_vertexOut);
    hipMalloc(&dev_vertexOut, vertCount * sizeof(VertexOut));

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo,glm::vec3 lightPos,glm::vec3 cameraUp,glm::vec3 cameraFront,float fovy,float cameraDis,float rotation) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);
	
    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	vertexShading<<<(vertCount+127)/128,128>>>(dev_bufVertex,dev_vertexOut,cameraUp,cameraFront,fovy,cameraDis,rotation,vertCount);
	primitiveAssemblyTest<<<(bufIdxSize/3+127)/128,128>>>(dev_vertexOut,dev_bufIdx,dev_primitives,bufIdxSize/3);
	setColorToBlack<<<(width*height+127)/128,128>>>(dev_depthbuffer,width*height);
	
	setDepthMax<<<(width*height+127)/128,128>>>(dev_depth,width*height);
	rasterization<<<(bufIdxSize/3+127)/128,128>>>(dev_primitives,dev_depthbuffer,dev_depth,lightPos,cameraFront,bufIdxSize/3,width);
	
    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_vertexOut);
	dev_vertexOut=NULL;

	hipFree(dev_depth);
	dev_depth=NULL;

    checkCUDAError("rasterizeFree");
}
