#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static int *d_mutex = NULL;
static VertexIn *dev_bufVertex = NULL;
static VertexOut *dev_vOut = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static glm::vec3 *d_lightSourcePos = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;
#define ANTIALIASING 1.0f
#define TWOAA 1.0f

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/*
// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}
*/

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
	hipMalloc(&dev_depthbuffer, TWOAA * width * height * sizeof(Fragment));
	hipMemset(dev_depthbuffer, 0, TWOAA * width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));

	hipMalloc(&d_mutex, TWOAA * width * height * sizeof(int));
	hipMemset(d_mutex, 0, TWOAA * width * height * sizeof(int));

    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
		bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	hipFree(dev_vOut);
	hipMalloc(&dev_vOut, vertCount * sizeof(VertexIn));

    hipFree(dev_primitives);
	hipMalloc(&dev_primitives, bufIdxSize / 3 * sizeof(Triangle));

	hipFree(d_lightSourcePos);
	hipMalloc(&d_lightSourcePos, sizeof(glm::vec3));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo, glm::mat4 viewProjecition) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

	clearBuffers << <blockCount2d, blockSize2d >> > 
		(dev_depthbuffer, width, height);
	checkCUDAError("clearDepthBuffer");

    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	glm::vec3 lightSource(0, 2, 0);
	hipMemcpy(d_lightSourcePos, &lightSource, sizeof(glm::vec3), hipMemcpyHostToDevice);

	int bSize = 64;
	int numBlock = ceil(((float)vertCount) / bSize);
	vertexShader << <numBlock, bSize >> > (dev_bufVertex, dev_vOut, vertCount, viewProjecition);
	checkCUDAError("vShader");

	int numTri = bufIdxSize / 3;
	numBlock = ceil((float)numTri / bSize);
	primitiveAssembly << <numBlock, bSize >> > (dev_bufVertex, dev_vOut, dev_bufIdx, numTri, dev_primitives);
	checkCUDAError("primitiveAssembly");

	//backface culling
	Triangle* new_end = thrust::remove_if(thrust::device, dev_primitives, dev_primitives + numTri, facing_backward());
	numTri = new_end - dev_primitives;

	glm::ivec2 scissorMin(0, 0);
	glm::ivec2 scissorMax(800, 800);

	rasterization << <numBlock, bSize >> > (dev_primitives, numTri,
		dev_depthbuffer, width, height, d_mutex, d_lightSourcePos, scissorMin, scissorMax);
	checkCUDAError("rasterization");

	copyToFrameBuffer << < blockCount2d, blockSize2d >> >(dev_framebuffer,
		dev_depthbuffer, width, height);
	checkCUDAError("copyToFrameBuffer");

	// rClr << < blockCount2d, blockSize2d >> >(dev_primitives, bufIdxSize / 3,
	//	dev_framebuffer, width, height);

    // Copy depthbuffer colors into framebuffer
    //render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_vOut);
	dev_vOut = NULL;

    checkCUDAError("rasterizeFree");
}


__global__ void clearBuffers(Fragment* dev_depthbuffer,
	int screenWidth, int screenHeight){

	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	if (i >= screenWidth || j >= screenHeight) return;

	int ptr = (j * TWOAA * screenWidth) + (i * TWOAA);
	for (int offset = 0; offset < TWOAA; offset++){
		dev_depthbuffer[ptr + offset].depth = INFINITY;
		dev_depthbuffer[ptr + offset].col = glm::vec3(0, 0, 0);
	}
}
//per vertex
__global__ void vertexShader(VertexIn* d_vertsIn, VertexOut* d_vertsOut, int vertsNum, 
	glm::mat4 viewProjection){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= vertsNum) return;

	VertexIn in = d_vertsIn[i];
	VertexOut out;

	glm::vec4 v = viewProjection * glm::vec4(in.pos, 1);
	out.pos = glm::vec3(v / v.w);

	d_vertsOut[i] = out;
}

//per triangle!
__global__ void primitiveAssembly(VertexIn* d_vertsIn, VertexOut* d_vertsOut, int* d_idx, int triangleNo, Triangle* d_tri){
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= triangleNo) return;

	Triangle out;
	out.vOut[0] = d_vertsOut[d_idx[(3*i)]];
	out.vOut[1] = d_vertsOut[d_idx[(3*i)+1]];
	out.vOut[2] = d_vertsOut[d_idx[(3*i)+2]];

	out.vIn[0] = d_vertsIn[d_idx[(3*i)]];
	out.vIn[1] = d_vertsIn[d_idx[(3*i)+1]];
	out.vIn[2] = d_vertsIn[d_idx[(3*i)+2]];

	d_tri[i] = out;
}

//perform rasterization per Triangle
__global__ void rasterization(Triangle* d_tri, int triNo,
	Fragment* dev_depthbuffer, int screenWidth, int screenHeight, int* mutex,
	glm::vec3 *lightSourcePos,
	glm::ivec2 scissorMin = glm::ivec2(0, 0), 
	glm::ivec2 scissorMax = glm::ivec2(width, height))
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= triNo) return;

	Triangle t = d_tri[i];
	glm::vec3 tri[3] = { t.vOut[0].pos, t.vOut[1].pos, t.vOut[2].pos };
	float signedAreaTri = calculateSignedArea(tri);
	if (signedAreaTri <= 0) return;

	AABB bbox = getAABBForTriangle(tri);

	if (bbox.max.z > 0 && bbox.min.z < -1) return;

	//start rasterizing from min to max
	//dont forget that the screen starts from -1 to 1
	int maxY = ceil((1 - bbox.min.y) * screenHeight / 2);
	if (maxY > scissorMax.y) maxY = scissorMax.y;

	int maxX = ceil((bbox.max.x + 1) * screenWidth / 2);
	if (maxX > scissorMax.x) maxX = scissorMax.x;

	int y = (1 - bbox.max.y) * screenHeight / 2;
	if (y < scissorMin.y) y = scissorMin.y;

	int minX = (bbox.min.x + 1) * screenWidth / 2;
	if (minX < scissorMin.x) minX = scissorMin.x;


	glm::vec2 p;
	for (; y < maxY; y++){
		for (int x = minX; x < maxX; x++){
			for (int k = 0; k < ANTIALIASING; k++){
				for (int l = 0; l < ANTIALIASING; l++){
					float offsetY = (0.5f / ANTIALIASING) + (1.0f / ANTIALIASING)*k;
					float offsetX = (0.5f / ANTIALIASING) + (1.0f / ANTIALIASING)*l;

					p.x = -1 + ((x + offsetX) / screenWidth * 2);
					p.y = 1 - ((y + offsetY) / screenHeight * 2);
					glm::vec3 bCoord = calculateBarycentricCoordinate(tri, p, signedAreaTri);

					if (isBarycentricCoordInBounds(bCoord)){
						float depth = getZAtCoordinate(bCoord, tri);

						int ptr = (y * TWOAA * screenWidth) + (x * TWOAA) +
							(k * ANTIALIASING) + l;

						// mutex code from stackOverflow
						// Loop-wait until this thread is able to execute its critical section.
						bool isSet;
						do {
							isSet = (atomicCAS(&mutex[ptr], 0, 1) == 0);
							if (isSet) {
								// Critical section goes here.
								// The critical section MUST be inside the wait loop;
								// if it is afterward, a deadlock will occur.

								if (depth < dev_depthbuffer[ptr].depth){
									dev_depthbuffer[ptr].depth = depth;
								
									glm::vec3 pos = (bCoord.x * t.vIn[0].pos) + (bCoord.y * t.vIn[1].pos) + (bCoord.z * t.vIn[2].pos);
									glm::vec3 clr = (bCoord.x * t.vIn[0].col) + (bCoord.y * t.vIn[1].col) + (bCoord.z * t.vIn[2].col);
									glm::vec3 nor = (bCoord.x * t.vIn[0].nor) + (bCoord.y * t.vIn[1].nor) + (bCoord.z * t.vIn[2].nor);

									dev_depthbuffer[ptr].col = glm::dot(glm::normalize(*lightSourcePos - pos), nor) * clr;
								}
							}
							if (isSet) {
								mutex[ptr] = 0;
							}
						} while (!isSet);
					}
				}
			}
		}
	}
}

__global__ void copyToFrameBuffer(glm::vec3* dev_framebuffer, Fragment* dev_depthbuffer, int width, int height){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int i = x + (y * width);

	if (x >= width || y >= height) return;

	glm::vec3 clrOut(0, 0, 0);

	int ptr = (y * TWOAA * width) + (x * TWOAA);
	for (int offset = 0; offset < TWOAA; offset++){
		if (dev_depthbuffer[ptr + offset].depth != INFINITY){
			clrOut += dev_depthbuffer[ptr + offset].col;
		}
		else 
			clrOut += glm::vec3(0.3, 0.3, 0.3);
	}
	
	dev_framebuffer[i] = clrOut / TWOAA;
}