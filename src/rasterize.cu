#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"



struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;

	glm::vec3 tex;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;

	glm::vec3 ndc;
	glm::vec3 winPos;

	glm::vec3 tex;
    // TODO
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
	int depth;
    glm::vec3 color;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
VertexOut *dev_bufVtxOut = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
glm::vec3 **dev_textures = NULL;
glm::vec2 * dev_texInfo = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;
static int bufTexSize = 0;
glm::mat4 M_win;
glm::mat4 M_view;

__global__
void kernBufInit(int w, int h, Fragment * depthbuffer, glm::vec3 *framebuffer)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) 
	{
		depthbuffer[index].depth =2* MAX_DEPTH; //INFINITY;//!!!
		depthbuffer[index].color = glm::vec3(0.2, 0, 0);
	}
}

__global__			//(vertCount,     glm::mat4() ,       M_view,          projMat,            dev_bufVertex, dev_bufVtxOut, M_win);
void kernVertexShader(int vtxCount,glm::mat4 M_model, glm::mat4 M_view, glm::mat4 M_Projection, VertexIn *vtxI, VertexOut *vtxO, glm::mat4 M_win)
{
	//demo:http://www.realtimerendering.com/udacity/transforms.html
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < vtxCount)
	{
		glm::vec4 P_clip = M_Projection * M_view * M_model * glm::vec4(vtxI[index].pos, 1);	//clip coords
		glm::vec4 P_NDC = P_clip*(1 / P_clip.w);//!!!w-divide for NDC	: P_clip/w
		//!!!window coords		: M_win*P_NDC
		
		vtxO[index].ndc = glm::vec3(P_NDC);
		//vtxO[index].ndc = vtxI[index].pos;
		vtxO[index].nor = vtxI[index].nor;
		vtxO[index].col = vtxI[index].col;
		P_NDC = M_win*P_NDC;
		vtxO[index].winPos = glm::vec3(P_NDC);
		vtxO[index].tex = vtxI[index].tex;
	}
}

__global__
void kernPrimitiveAssembly(Triangle* primitives,int* bufIdx,int bufIdxSize, VertexOut * bufVtxOut)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index<bufIdxSize/3)
	{
		int i = 3 * index;
		primitives[index].v[0] = bufVtxOut[bufIdx[i]];
		primitives[index].v[1] = bufVtxOut[bufIdx[i+1]];
		primitives[index].v[2] = bufVtxOut[bufIdx[i+2]];
	}
}

__host__ __device__ glm::vec3 ColorInTex(int texId, glm::vec3**texs, glm::vec2*info, glm::vec2 uv)
{
	int xSize = info[texId].x;
	int ySize = info[texId].y;
	if (uv.x < 0 || uv.y < 0 || uv.x >1 || uv.y >1) return glm::vec3(0, 0, 0);
	float u = (float)(uv.x*(float)xSize);
	float v = (float)(uv.y*(float)ySize);
	int k = u;
	int j = v;
	//if (k == 0 || k == xSize - 1 || j == 0 || j == ySize - 1)//!!!border
	if (true)
	{
		return texs[texId][(j * xSize) + k];
	}
	//else return glm::vec3(0, 0, 0);
	//bilinear filtering - within
	/*
	//https://en.wikipedia.org/wiki/Bilinear_interpolation
	if (x < (float)k + 0.5)	//left part
	k -= 1;
	if (y < (float)j + 0.5)//left top
	j -= 1;

	//k,k+1
	//j,j+1
	glm::vec3 p11 = texs[texId][(j * xSize) + k];
	glm::vec3 p12 = texs[texId][(j * xSize) + k + 1];
	glm::vec3 p21 = texs[texId][((j + 1) * xSize) + k];
	glm::vec3 p22 = texs[texId][((j + 1) * xSize) + k + 1];

	float x1 = k;
	float x2 = k + 2;
	float y1 = j;
	float y2 = j + 2;

	glm::vec3 f_xy1 = p11*(x2 - x) / (x2 - x1) + p21*(x - x1) / (x2 - x1);
	glm::vec3 f_xy2 = p12*(x2 - x) / (x2 - x1) + p22*(x - x1) / (x2 - x1);

	glm::vec3 f_xy = f_xy1*(y2 - y) / (y2 - y1) + f_xy2*(y - y1) / (y2 - y1);
	return f_xy;*/

	//bilnear - 9 pixels
	/*
	glm::vec3 t[3][3];
	for (int m = 0; m < 3; m++)
	{
		for (int n = 0; n < 3; n++)
		{
			t[m][n] = texs[texId][((j + m - 1) * xSize) + (k + n - 1)];
		}

	}

	glm::vec3 p11 = (t[0][0] + t[0][1] + t[1][0] + t[1][1])*0.25f;
	glm::vec3 p12 = (t[0][1] + t[0][2] + t[1][1] + t[1][1])*0.25f;
	glm::vec3 p21 = (t[1][0] + t[1][1] + t[2][0] + t[2][1])*0.25f;
	glm::vec3 p22 = (t[1][1] + t[1][2] + t[2][1] + t[2][2])*0.25f;

	float x1 = k;
	float x2 = k + 1;
	float y1 = j;
	float y2 = j + 1;
	float c1 = (x2 - x) / (x2 - x1);
	float c2 = (x - x1) / (x2 - x1);
	float c3 = (y2 - y) / (y2 - y1);
	float c4 = (y - y1) / (y2 - y1);

	glm::vec3 f_xy1 = p11*c1 + p21*c2;
	glm::vec3 f_xy2 = p12*c1 + p22*c2;

	glm::vec3 f_xy = f_xy1*c3 + f_xy2*c4;
	return f_xy;*/

}

__host__ __device__ glm::vec3 ColorInTexBilinear(int texId, glm::vec3**texs, glm::vec2*info, glm::vec2 uv)
{
	//https://en.wikipedia.org/wiki/Bilinear_filtering
	int xSize = info[texId].x;
	int ySize = info[texId].y;
	if (uv.x < 0 || uv.y < 0 || uv.x >1 || uv.y >1) return glm::vec3(0, 0, 0);
	float u = (float)(uv.x*(float)xSize - 0.5);
	float v = (float)(uv.y*(float)ySize - 0.5);

	//u = u * tex.size - 0.5;
	//v = v * tex.size - 0.5;
	int x = floor(u);
	int y = floor(v);
	float u_ratio = u - x;
	float v_ratio = v - y;
	float u_opposite = 1 - u_ratio;
	float v_opposite = 1 - v_ratio;

	texs[texId][(y * xSize) + x];
	if (x == 0 || x == xSize - 1 || y == 0 || y == ySize - 1)//!!!border
	{
		return texs[texId][(y * xSize) + x];
	}
	glm::vec3 result = (texs[texId][(y * xSize) + x] * u_opposite + texs[texId][(y * xSize) + x + 1] * u_ratio) * v_opposite +
		(texs[texId][((y + 1) * xSize) + x] * u_opposite + texs[texId][((y + 1) * xSize) + x + 1] * u_ratio) * v_ratio;
	return result;
}

__global__
void kernRasterizer(int w, int h, Fragment * depthbuffer, Triangle*primitives, int bufIdxSize, glm::vec3 lightWorld, glm::mat4 allMat, glm::vec3** texs, glm::vec2* tInfo)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (index < bufIdxSize / 3)
	{
		//Scanline
		glm::vec3 tri[3];
		tri[0] = primitives[index].v[0].winPos;
		tri[1] = primitives[index].v[1].winPos;
		tri[2] = primitives[index].v[2].winPos;

		glm::vec3 tex[3];
		tex[0] = primitives[index].v[0].tex;
		tex[1] = primitives[index].v[1].tex;
		tex[2] = primitives[index].v[2].tex;
		//!!! currently linear . later interpolation
		glm::vec3 normal = glm::normalize(primitives[index].v[0].nor + primitives[index].v[1].nor + primitives[index].v[2].nor);
		glm::vec3 color = glm::normalize(primitives[index].v[0].col + primitives[index].v[1].col + primitives[index].v[2].col);

		/*
		//http://keisan.casio.com/exec/system/1223596129
		glm::vec3 A = primitives[index].v[0].ndc;
		glm::vec3 B = primitives[index].v[1].ndc;
		glm::vec3 C = primitives[index].v[2].ndc;

		float a = (B.y - A.y)*(C.z - A.z) - (C.y - A.y)*(B.z - A.z);
		float b = (B.z - A.z)*(C.x - A.x) - (C.z - A.z)*(B.x - A.x);
		float c = (B.x - A.x)*(C.y - A.y) - (C.x - A.x)*(B.y - A.y);
		float d = -(a*A.x + b*A.y + c*A.x);
		//ax+by+cz+d = 0;
		*/
		AABB triBox = getAABBForTriangle(tri);
		for (int x = triBox.min.x; x <= triBox.max.x; x++)
		{
			for (int y = triBox.min.y; y <= triBox.max.y; y++)
			{
				glm::vec3 bPoint = calculateBarycentricCoordinate(tri, glm::vec2(x, y));
				//!!! later line segment
				if (isBarycentricCoordInBounds(bPoint)) // Inside triangle
				{
					//glm::vec4 crntNDC = glm::inverse(M_win)*glm::vec4(x, y, 1,1);
					//crntNDC.z = (a*crntNDC.x + b*crntNDC.y + d) / (-c);
					//crntNDC = M_win*crntNDC;
					//int crntDepth = (int)(tri[0].z * 1000);
					//int crntDepth = (int)(crntNDC.z * 1000);
					//!!! later clipping
					if (x<0 || x>w || y<0 || y>h)
						continue;
					float crntDepth = getZAtCoordinate(bPoint, tri);
					crntDepth *= MAX_DEPTH;
					int orig = atomicMin(&(depthbuffer[x+y*w].depth), (int)crntDepth);
					if (orig >= crntDepth)
					//if (depthbuffer[x + y*w].depth==crntDepth)
					{
						glm::vec3 Pos = tri[0] * bPoint.x + tri[1] * bPoint.y + tri[2] * bPoint.z;
						glm::vec3 uv = tex[0] * bPoint.x + tex[1] * bPoint.y + tex[2] * bPoint.z;
						//texture mapping !!! later : repeat, offset...
						if (texs != NULL &&tInfo != NULL)
							color = ColorInTexBilinear(0, texs, tInfo, glm::vec2(uv));
						glm::vec4 PosWorld = glm::inverse(allMat)* glm::vec4(Pos, 1);
						glm::vec3 lightDir = glm::normalize(lightWorld - glm::vec3(PosWorld));
						float diffuse = max(dot(lightDir, normal), 0.0);
						depthbuffer[x + y*w].color =  color*diffuse;
						//depthbuffer[x + y*w].color = normal;
					}
				}
			}
		}

	}
}

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;

	int hWidth = width / 2;
	int hHeight = height / 2;
	M_win = glm::mat4(\
		hWidth, 0, 0, 0, \
		0, hHeight, 0, 0, \
		0, 0, 0.5, 0,
		hWidth, hHeight, 0.5, 1
		);

    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
	//hipMemset(dev_depthbuffer, INFINITY, width * height * sizeof(Fragment));

    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers( obj * mesh ) {
  
	bufIdxSize = mesh->getBufIdxsize();
	int *bufIdx = mesh->getBufIdx();
	bufTexSize = mesh->getBufTexsize() / 3;
	float * bufTex = mesh->getBufTex();
	vertCount = mesh->getBufPossize() / 3;
	float *bufPos = mesh->getBufPos();
	float *bufNor = mesh->getBufNor();
	float *bufCol = mesh->getBufCol();

	//Copy materials to dev_textures
	
	int texSize = mesh->textureImages.size()*sizeof(glm::vec3 *);
	int texInfoSize = mesh->textureImages.size()*sizeof(glm::vec2);
	if (texSize > 0 && texInfoSize > 0)
	{
		hipMalloc((void**)&dev_textures, texSize);
		hipMalloc((void**)&dev_texInfo, texInfoSize);
		std::vector<glm::vec3*> tempImg;
		std::vector<glm::vec2> tempInfo;
		for (int i = 0; i < mesh->textureImages.size(); i++)
		{
			glm::vec3 * dev_img;
			int imgSize = mesh->textureImages[i].getSize()*sizeof(glm::vec3);
			hipMalloc((void**)&dev_img, imgSize);
			hipMemcpy(dev_img, mesh->textureImages[i].pixels, imgSize, hipMemcpyHostToDevice);
			tempImg.push_back(dev_img);
			tempInfo.push_back(glm::vec2(mesh->textureImages[i].xSize, mesh->textureImages[i].ySize));
		}
		hipMemcpy(dev_textures, tempImg.data(), texSize, hipMemcpyHostToDevice);
		hipMemcpy(dev_texInfo, tempInfo.data(), texInfoSize, hipMemcpyHostToDevice);
	}

	
	//
    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

	VertexIn *bufVertex = new VertexIn[vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
		bufVertex[i].tex = glm::vec3(bufTex[j + 0], bufTex[j + 1], bufTex[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	//!!!
	hipFree(dev_bufVtxOut);
	hipMalloc(&dev_bufVtxOut, vertCount * sizeof(VertexOut));

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo,glm::mat4 viewMat,glm::mat4 projMat) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

	M_view = viewMat;//glm::lookAt(eye, center, up);
    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	int bSize_vtx = 128;
	int bSize_pri = 128;
	dim3 gSize_vtx((vertCount + bSize_vtx - 1) / bSize_vtx);
	dim3 gSize_pri((bufIdxSize/3 + bSize_pri - 1) / bSize_pri);

	glm::vec4 light(0.3, 0.4, 0.5,1);
	//glm::vec4 lightWin = M_win*projMat * M_view * glm::mat4() *light;
	glm::vec4 lightW = light;// projMat * M_view * glm::mat4() *light;
	glm::mat4 M_all = M_win*projMat * M_view * glm::mat4();
	//****** 1. Clear depth buffer
	kernBufInit << <blockCount2d, blockSize2d >> >(width, height, dev_depthbuffer, dev_framebuffer);
	//****** 2. Vertex Shading
	//	VertexIn[n] vs_input -> VertexOut[n] vs_output

	kernVertexShader << <gSize_vtx, bSize_vtx >> >(vertCount, glm::mat4(), M_view, projMat, dev_bufVertex, dev_bufVtxOut, M_win );

	VertexOut * textVtxOut = new VertexOut[vertCount];
	hipMemcpy(textVtxOut, dev_bufVtxOut, vertCount*sizeof(VertexOut), hipMemcpyDeviceToHost);
	/*for (int i = 0; i < vertCount; i++)
	{
		glm::vec3 temp = textVtxOut[i].ndc;
		printf("tri[%d] after VtxShader:%2f,%2f,%2f\n",i,temp.x,temp.y,temp.z);
	}*/

	//****** 3. Primitive Assembly
	//  VertexOut[n] vs_output -> Triangle[n/3] primitives
	kernPrimitiveAssembly<<<gSize_pri, bSize_pri >>>(dev_primitives, dev_bufIdx, bufIdxSize, dev_bufVtxOut);

	//****** 4. Rasterization
	//  Triangle[n/3] primitives -> FragmentIn[m] fs_input
	kernRasterizer << <gSize_pri, bSize_pri >> >(width, height, dev_depthbuffer, dev_primitives, bufIdxSize, glm::vec3(lightW),M_all,dev_textures,dev_texInfo);
	//****** 5. Fragment shading
	//****** 6. Fragments to depth buffer
	//****** 7. Depth buffer for storing & testing fragments
	//****** 8. Fragment to framebuffer writing

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {

	hipFree(dev_bufVtxOut);
	dev_bufVtxOut = NULL;

    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

	hipFree(dev_textures);
	hipFree(dev_texInfo);

    checkCUDAError("rasterizeFree");
}
