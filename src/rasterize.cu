#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
    // TODO
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)



	//so each stage of the pipeline is just anotehr kernel function?




    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
