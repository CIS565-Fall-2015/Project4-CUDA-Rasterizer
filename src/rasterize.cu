#include "hip/hip_runtime.h"
/*
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <climits>
#include <hip/hip_runtime.h>

#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/remove.h>

#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>
#include <glm/gtx/transform.hpp>

#include <util/utilityCore.hpp>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"
#include "sceneStructs.h"

/************************* Struct Definitions *********************************/

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
};
struct VertexOut {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;

    glm::vec3 worldPos;
};
struct Triangle {
    glm::vec3 pos[3];
    glm::vec3 nor[3];
    glm::vec3 col[3];

    glm::vec3 worldPos[3];
    bool valid;
    bool middle;
};
struct Fragment {
    glm::vec3 color;
    Triangle tri;
    glm::vec3 baryCoords;
    int z;
    bool valid;
};

static float t = 0;

static int width = 0;
static int height = 0;
static int bufIdxSize = 0;
static int vertCount = 0;
static int primMultFactor = 4;

static int       *dev_bufIdx         = NULL;
static VertexIn  *dev_bufVertexIn    = NULL;
static VertexOut *dev_bufVertexOut   = NULL;
static Triangle  *dev_origPrimitives = NULL;
static Triangle  *dev_genPrimitives  = NULL;
static Fragment  *dev_depthbuffer    = NULL;
static glm::vec3 *dev_framebuffer    = NULL;

__device__ VertexOut transformVertex(glm::vec3 pos, glm::vec3 nor,
        glm::mat4 model, glm::mat4 invModel, glm::mat4 mvp) {
    VertexOut vo;
    vo.pos = pos;
    vo.worldPos = multiplyMV(model, glm::vec4(pos, 1));
    vo.pos = multiplyMV(mvp, glm::vec4(pos, 1));
    vo.nor = glm::vec3(invModel * glm::vec4(nor, 0));
    vo.col = glm::vec3(0.f);
    return vo;
}

__device__ Triangle buildTriangle(VertexOut v0, VertexOut v1, VertexOut v2) {
    Triangle tri;
    tri.pos[0] = v0.pos;
    tri.pos[1] = v1.pos;
    tri.pos[2] = v2.pos;

    tri.nor[0] = v0.nor;
    tri.nor[1] = v1.nor;
    tri.nor[2] = v2.nor;

    tri.col[0] = v0.col;
    tri.col[1] = v1.col;
    tri.col[2] = v2.col;

    tri.worldPos[0] = v0.worldPos;
    tri.worldPos[1] = v1.worldPos;
    tri.worldPos[2] = v2.worldPos;
    tri.valid = true;
    tri.middle = false;
    return tri;
}

__device__ void printVec3(glm::vec3 v) {
    printf("(%f, %f, %f)\n", v.x, v.y, v.z);
}

__device__ void printMat4(const glm::mat4 &m) {
    printf("%f, %f, %f, %f\n", m[0][0], m[1][0], m[2][0], m[3][0]);
    printf("%f, %f, %f, %f\n", m[0][1], m[1][1], m[2][1], m[3][1]);
    printf("%f, %f, %f, %f\n", m[0][2], m[1][2], m[2][2], m[3][2]);
    printf("%f, %f, %f, %f\n", m[0][3], m[1][3], m[2][3], m[3][3]);
}

/************************* Output to Screen ***********************************/

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/**
 * Writes fragment colors to the framebuffer
 */
__global__ void render(int w, int h, Fragment *depthbuffer,
        glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < w && y < h) {
        int frameidx = x + (y * w);
        int depthidx = 4*frameidx;
        glm::vec3 color = depthbuffer[depthidx].color +
            depthbuffer[depthidx+1].color +
            depthbuffer[depthidx+2].color +
            depthbuffer[depthidx+3].color;
        framebuffer[frameidx] = color / 4.f;
    }
}

/************************* Initialization *************************************/

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;

    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,    4*width * height * sizeof(Fragment));
    hipMemset( dev_depthbuffer, 0, 4*width * height * sizeof(Fragment));

    hipFree(dev_bufVertexOut);
    hipMalloc(&dev_bufVertexOut,    width * height * sizeof(VertexOut));
    hipMemset( dev_bufVertexOut, 0, width * height * sizeof(VertexOut));

    hipFree(dev_genPrimitives);
    hipMalloc(&dev_genPrimitives,    primMultFactor * width * height * sizeof(Triangle));
    hipMemset( dev_genPrimitives, 0, primMultFactor * width * height * sizeof(Triangle));

    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,    width * height * sizeof(glm::vec3));
    hipMemset( dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertexIn = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertexIn[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertexIn[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertexIn[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertexIn);
    hipMalloc(&dev_bufVertexIn, vertCount * sizeof(VertexIn));
    hipMemcpy( dev_bufVertexIn, bufVertexIn, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_origPrimitives);
    hipMalloc(&dev_origPrimitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_origPrimitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/************************* Rasterization Pipeline *****************************/

__device__ void clearFragment(int idx, Fragment *depthbuffer) {
    depthbuffer[idx].valid = false;
    depthbuffer[idx].z = INT_MAX;
    depthbuffer[idx].color = glm::vec3(.15, .15, .15);
}

__global__ void clearDepthBuffer(int width, int height, Fragment *depthbuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < width && y < height) {
        int index = x + (y * width);
        clearFragment(4*index  , depthbuffer);
        clearFragment(4*index+1, depthbuffer);
        clearFragment(4*index+2, depthbuffer);
        clearFragment(4*index+3, depthbuffer);
    }
}

// Applies vertex transformations (from given model-view-projection matrix)
__global__ void vertexShader(int vertcount,
        VertexIn *verticesIn, VertexOut *verticesOut,
        glm::mat4 model, glm::mat4 invModel, glm::mat4 mvp) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < vertcount) {
        VertexIn vin = verticesIn[k];

        //VertexOut vo = transformVertex(vin.pos, vin.nor, model, invModel, mvp);
        VertexOut vo;
        vo.pos = vin.pos;
        vo.nor = vin.nor;
        verticesOut[k] = vo;
    }
}

// Assembles sets of 3 vertices into Triangles.
__global__ void assemblePrimitives(int primitivecount, VertexOut *vertices,
        int *indices, Triangle *primitives) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < primitivecount) {
        VertexOut v0 = vertices[indices[k*3  ]];
        VertexOut v1 = vertices[indices[k*3+1]];
        VertexOut v2 = vertices[indices[k*3+2]];
        primitives[k] = buildTriangle(v0, v1, v2);
    }
}

__device__ void subdivide(int idx, Triangle tri, Triangle *genprimitives,
        glm::mat4 model, glm::mat4 invModel, glm::mat4 mvp) {
    // New vertices
    glm::vec3 v0pos = tri.pos[0];
    glm::vec3 v1pos = tri.pos[1];
    glm::vec3 v2pos = tri.pos[2];
    glm::vec3 v3pos = midpoint(v0pos, v1pos);
    glm::vec3 v4pos = midpoint(v0pos, v2pos);
    glm::vec3 v5pos = midpoint(v1pos, v2pos);

    // New normals
    glm::vec3 v0nor = tri.nor[0];
    glm::vec3 v1nor = tri.nor[1];
    glm::vec3 v2nor = tri.nor[2];
    glm::vec3 v3nor = midpoint(v0nor, v1nor);
    glm::vec3 v4nor = midpoint(v0nor, v2nor);
    glm::vec3 v5nor = midpoint(v1nor, v2nor);

    VertexOut v0 = transformVertex(v0pos, v0nor, model, invModel, mvp);
    VertexOut v1 = transformVertex(v1pos, v1nor, model, invModel, mvp);
    VertexOut v2 = transformVertex(v2pos, v2nor, model, invModel, mvp);
    VertexOut v3 = transformVertex(v3pos, v3nor, model, invModel, mvp);
    VertexOut v4 = transformVertex(v4pos, v4nor, model, invModel, mvp);
    VertexOut v5 = transformVertex(v5pos, v5nor, model, invModel, mvp);

    genprimitives[idx  ] = buildTriangle(v0, v3, v4);
    genprimitives[idx+1] = buildTriangle(v3, v1, v5);
    genprimitives[idx+2] = buildTriangle(v4, v5, v2);
    genprimitives[idx+3] = buildTriangle(v3, v5, v4); // middle triangle
    genprimitives[idx+3].middle = true;
}

__global__ void geometryShader(int primitivecount, int multFactor, Triangle *primitives,
        Triangle *genprimitives, glm::vec3 eye,
        glm::mat4 model, glm::mat4 invModel, glm::mat4 mvp) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < primitivecount) {
        int genidx = k * multFactor;

        // Back-face culling
        Triangle tri = primitives[k];
        glm::vec3 n = glm::cross(tri.pos[1] - tri.pos[0], tri.pos[2] - tri.pos[0]);
        float dir = glm::dot(eye - tri.pos[0], n);
        subdivide(genidx, tri, genprimitives, model, invModel, mvp);
        if (dir < 0.f) {
            // Triangle tessellation
            //subdivide(genidx, tri, genprimitives);
        } else {
            //subdivide(genidx, tri, genprimitives);
            //genprimitives[genidx  ].valid = false;
            //genprimitives[genidx+1].valid = false;
            //genprimitives[genidx+2].valid = false;
            //genprimitives[genidx+3].valid = false;
        }
    }
}

__device__ void storeFragment(float x, float y, float width, float height,
        int fragmentidx, Triangle tri, Fragment *fragments) {

    glm::vec3 bary = calculateBarycentricCoordinate(tri.pos, glm::vec2(x, y));

    if (isBarycentricCoordInBounds(bary)) {
        Fragment prev = fragments[fragmentidx];

        float z = getZAtCoordinate(tri.worldPos, bary);
        int depth = z * INT_MAX;
        atomicMin(&fragments[fragmentidx].z, depth);

        if (fragments[fragmentidx].z == depth) {
            fragments[fragmentidx] = (Fragment) { glm::vec3(0, 0, 0), tri, bary, depth, true};
        }
    } else {
    }
}

// Scans across triangles to generate primitives (pixels).
__global__ void scanline(int w, int h, int tricount,
        Triangle *primitives, Fragment *fragments) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < tricount) {
        Triangle tri = primitives[k];
        if (tri.valid == false) {
            return;
        }

        float ystep = 2.f / h;
        float xstep = 2.f / w;

        float yjit = ystep / 4;
        float xjit = xstep / 4;

        AABB bb = getAABBForTriangle(tri.pos);

        float ymin = glm::max(-1.f, (int) (bb.min.y / ystep) * ystep);
        float xmin = glm::max(-1.f, (int) (bb.min.x / xstep) * xstep);
        float ymax = glm::min(1.f, bb.max.y);
        float xmax = glm::min(1.f, bb.max.x);
        for (float y = ymin; y < ymax; y += ystep) {
            for (float x = xmin; x < xmax; x += xstep) {
                glm::vec2 pos = fromNDC(x, y, w, h);
                int fragmentidx = 4*(pos.x + (pos.y * w));

                storeFragment(x,      y,      w, h, fragmentidx,   tri, fragments);
                storeFragment(x+xjit, y,      w, h, fragmentidx+1, tri, fragments);
                storeFragment(x,      y+yjit, w, h, fragmentidx+2, tri, fragments);
                storeFragment(x+xjit, y+yjit, w, h, fragmentidx+3, tri, fragments);
            }
        }
    }
}

__device__ void colorFragment(Fragment &frag, glm::vec3 light) {
    if (frag.valid) {
        glm::vec3 norm = barycentricInterpolate(frag.tri.nor, frag.baryCoords);
        glm::vec3 pos = barycentricInterpolate(frag.tri.worldPos, frag.baryCoords);
        glm::vec3 lightdir = glm::normalize(light - pos);
        frag.color = glm::dot(lightdir, norm) * glm::vec3(1, 0, 0);
        frag.color = glm::abs(glm::normalize(norm));
        if (frag.tri.middle == true) {
            frag.color *= .75;
        }
    }
}

__global__ void fragmentShader(int width, int height,
        Fragment *fragments, glm::vec3 light) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < width && y < height) {
        int index = 4*(x + (y * width));
        colorFragment(fragments[index]  , light);
        colorFragment(fragments[index+1], light);
        colorFragment(fragments[index+2], light);
        colorFragment(fragments[index+3], light);
    }
}

struct terminator {
    __device__ bool operator()(const Triangle tri) {
        return tri.valid == false;
    }
};

int compactPrimitives(int primitivecount, Triangle *primitives) {
    Triangle *new_end = thrust::remove_if(thrust::device,
            primitives, primitives+primitivecount, terminator());
    return (new_end - primitives);
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    //t += 0.01f;

    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

    int tricount = bufIdxSize / 3;

    int sideLength1d = 16;
    dim3 blockSize1d(sideLength1d);
    dim3 vertBlockCount((vertCount + sideLength1d - 1) / sideLength1d);
    dim3 triBlockCount((tricount + sideLength1d - 1) / sideLength1d);

    Camera c;
    // Suzanne
//    c.position = glm::vec3(1, 1, 4);
//    c.view = glm::vec3(0, 1, 0);
//    c.up = glm::vec3(0, -1, 0);
//    c.light = glm::vec3(2, 5, -1);
//    c.fovy = glm::radians(45.f);

    // Cow
    c.position = glm::vec3(0, .2, -0.5);
    c.view = glm::vec3(0, .2, 0);
    c.up = glm::vec3(0, 1, 0);
    c.light = glm::vec3(0, 4, 5);
    c.fovy = 17.f;

    // Cube
//    c.position = glm::vec3(0, 1, 1);
//    c.view = glm::vec3(0, 0, 0);
//    c.up = glm::vec3(0, 1, 0);
//    c.light = glm::vec3(0, 4, 5);
//    c.fovy = glm::radians(40.f);

    glm::mat4 model = glm::rotate(t, glm::vec3(0.f, 1.f, 0.f));
    glm::mat4 invModel = glm::inverseTranspose(model);
    glm::mat4 view = glm::lookAt(c.position, c.view, c.up);
    glm::mat4 persp = glm::perspective(c.fovy, 1.f, 1.f, 10.f);
    glm::mat4 mvp = persp * view * model;

    // Set CudaEvents
    float vShadeTime, assPrimitivesTime, scanlineTime, fShadeTime;
    hipEvent_t begin, end;
    hipEventCreate(&begin);
    hipEventCreate(&end);

    // Clear Depth Buffer
    clearDepthBuffer<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);

    // VertexIn -> VertexOut
        hipEventRecord(begin);
    vertexShader<<<vertBlockCount, blockSize1d>>>(vertCount, dev_bufVertexIn,
            dev_bufVertexOut, model, invModel, mvp);
        checkCUDAError("");

        hipEventRecord(end); hipEventSynchronize(end); hipEventElapsedTime(&vShadeTime, begin, end);

    // VertexOut -> Triangle
        hipEventRecord(begin);
    assemblePrimitives<<<triBlockCount, blockSize1d>>>(tricount,
            dev_bufVertexOut, dev_bufIdx, dev_origPrimitives);
        checkCUDAError("");

        hipEventRecord(end); hipEventSynchronize(end); hipEventElapsedTime(&assPrimitivesTime, begin, end);

    // Triangle -> Triangle
        hipEventRecord(begin);
    geometryShader<<<triBlockCount, blockSize1d>>>(tricount, primMultFactor,
            dev_origPrimitives, dev_genPrimitives,
            c.position, model, invModel, mvp);
        checkCUDAError("");

        hipEventRecord(end); hipEventSynchronize(end); hipEventElapsedTime(&assPrimitivesTime, begin, end);

    int genPrimitiveCount = compactPrimitives(tricount * primMultFactor, dev_genPrimitives);
    dim3 genPrimCount((genPrimitiveCount + sideLength1d - 1) / sideLength1d);

    // Triangle -> Fragment
        hipEventRecord(begin);
    scanline<<<genPrimitiveCount, blockSize1d>>>(width, height, tricount *
            primMultFactor, dev_genPrimitives, dev_depthbuffer);
        checkCUDAError("");

        hipEventRecord(end); hipEventSynchronize(end); hipEventElapsedTime(&scanlineTime, begin, end);

    // Fragment -> Fragment
        hipEventRecord(begin);
    fragmentShader<<<blockCount2d, blockSize2d>>>(width, height,
            dev_depthbuffer, c.light);
        checkCUDAError("");

        hipEventRecord(end); hipEventSynchronize(end); hipEventElapsedTime(&fShadeTime, begin, end);

    // Clear CudaEvents
    hipEventDestroy(begin);
    hipEventDestroy(end);

    //fprintf(stderr, "%f %f %f %f\n", vShadeTime, assPrimitivesTime, scanlineTime, fShadeTime);

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertexOut);
    dev_bufVertexOut = NULL;

    hipFree(dev_bufVertexIn);
    dev_bufVertexIn = NULL;

    hipFree(dev_origPrimitives);
    dev_origPrimitives = NULL;

    hipFree(dev_genPrimitives);
    dev_genPrimitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
