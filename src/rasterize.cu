#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

//#include "sceneStructs.h"
#include "Scene.h"

extern Scene *scene;

struct keep
{
	__host__ __device__ bool operator()(const Triangle t)
	{
		return (!t.keep);
	}
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;
static glm::mat4 matrix;
static glm::vec3 camDir;
static Light light;
static Camera cam;

//Things added
static VertexOut *dev_outVertex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

__global__
void kernVertexShader(int numVertices, int w, int h, VertexIn * inVertex, VertexOut *outVertex, glm::mat4 matrix, glm::mat4 modelMat)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numVertices)
	{
		glm::vec4 outPoint = glm::vec4(inVertex[index].pos.x, inVertex[index].pos.y, inVertex[index].pos.z, 1.0f);

		outPoint = matrix * outPoint;

		if(outPoint.w != 0)
			outPoint /= outPoint.w;

		//In NDC
//		outVertex[index].pos = glm::vec3(outPoint);

		//In Device Coordinates
		outVertex[index].pos.x = outPoint.x * w;
		outVertex[index].pos.y = outPoint.y * h;
		outVertex[index].pos.z = outPoint.z;

		outVertex[index].nor = multiplyMV(modelMat, glm::vec4(inVertex[index].nor, 1.0f));
//		outVertex[index].col = glm::vec3(0,0,1);
//		outVertex[index].nor = inVertex[index].nor;

//		printf ("InVertex : %f %f \nOutVertex : %f %f \n\n", inVertex[index].pos.x, inVertex[index].pos.y, outVertex[index].pos.x, outVertex[index].pos.y);
	}
}

__global__
void kernPrimitiveAssembly(int numTriangles, VertexOut *outVertex, VertexIn *inVertex, Triangle *triangles, int* indices, glm::vec3 camDir)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		int k_3 = 3 * index;

		Triangle &t = triangles[index];
		glm::vec3 triNor = glm::normalize(inVertex[k_3].nor + inVertex[k_3+1].nor + inVertex[k_3+2].nor);

//		printf ("Tri Normal : %f %f %f\n", triNor.x, triNor.y, triNor.z);
//		printf ("Cam Dir : %f %f %f\n", camDir.x, camDir.y, camDir.z);

//		if(glm::dot(triNor, camDir) > -0.0001f)
//		{
//			t.keep = false;
//		}
//
//		else
		{
			t.keep = true;

			t.vOut[0] = outVertex[indices[k_3]];
			t.vOut[1] = outVertex[indices[k_3+1]];
			t.vOut[2] = outVertex[indices[k_3+2]];

			t.vIn[0] = inVertex[indices[k_3]];
			t.vIn[1] = inVertex[indices[k_3+1]];
			t.vIn[2] = inVertex[indices[k_3+2]];
		}
	}
}

__global__
void kernDrawAxis(int w, int h, Fragment *fragments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
    {
		int index = x + (y * w);
		if((x - w*0.5f) == 0)
	    {
			fragments[index].color = glm::vec3(0, 1, 0);
	    }
		else if((y - h*0.5f) == 0)
		{
			fragments[index].color = glm::vec3(1, 0, 0);
		}
    }
}

__global__
void kernClearFragmentBuffer(int w, int h, Fragment *fragments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < w && y < h)
    {
		int index = x + (y * w);

		fragments[index].color = glm::vec3(0, 0, 0);
		fragments[index].depth = INT_MAX;
    }
}

__global__
void kernRasterize(int w, int h, Fragment *fragments, Triangle *triangles, int numTriangles, Camera cam, Light light1, Light light2)
{
	//Rasterization per triangle
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		Triangle &t = triangles[index];

		glm::vec3 tri[3];
		tri[0] = t.vOut[0].pos;
		tri[1] = t.vOut[1].pos;
		tri[2] = t.vOut[2].pos;

		AABB aabb = getAABBForTriangle(tri);
		glm::ivec3 min, max;

		//Attempted clipping
		min.x = glm::clamp(aabb.min.x, -(float)w*0.5f+1, (float)w*0.5f-1);
		min.y = glm::clamp(aabb.min.y, -(float)h*0.5f+1, (float)h*0.5f-1);
		max.x = glm::clamp(aabb.max.x, -(float)w*0.5f+1, (float)w*0.5f-1);
		max.y = glm::clamp(aabb.max.y, -(float)h*0.5f+1, (float)h*0.5f-1);

		for(int i=min.x-1; i<=max.x+1; ++i)
		{
			for(int j=min.y-1; j<=max.y+1; ++j)
			{
				glm::ivec2 point(i,j);
				glm::vec3 barycentric = calculateBarycentricCoordinate(tri, point);

				if(isBarycentricCoordInBounds(barycentric))
				{
					glm::vec3 triIn[3];
					VertexIn tvIn[3] = {t.vIn[0], t.vIn[1], t.vIn[2]};

					triIn[0] = tvIn[0].pos;
					triIn[1] = tvIn[1].pos;
					triIn[2] = tvIn[2].pos;

					glm::vec3 norm = barycentric.x * tvIn[0].nor +
										barycentric.y * tvIn[1].nor +
						                barycentric.z * tvIn[2].nor;

					glm::vec3 pos = barycentric.x * tvIn[0].pos +
										barycentric.y * tvIn[1].pos +
										barycentric.z * tvIn[2].pos;

					glm::vec3 col = barycentric.x * tvIn[0].col +
										barycentric.y * tvIn[1].col +
										barycentric.z * tvIn[2].col;

					glm::vec3 lightVector1 = glm::normalize(light1.pos - pos);
					glm::vec3 lightVector2 = glm::normalize(light2.pos - pos);
					//glm::vec3 camVector = glm::normalize(cam.pos - pos);

					float diffusedTerm1 = glm::dot(lightVector1, norm);
					float diffusedTerm2 = glm::dot(lightVector2, norm);

//					if(diffusedTerm1 >0.0f || diffusedTerm2 > 0.0f)
					{
						int fragIndex = int((i+w*0.5) + (j + h*0.5)*w);
						int depth = getZAtCoordinate(barycentric, triIn) * 10000;

						//TODO : Use cuda atomics to avoid race condition here
						if(depth < fragments[fragIndex].depth)
						{
							atomicMin(&fragments[fragIndex].depth, depth);
							if(diffusedTerm1 > 0.0f && diffusedTerm2 > 0.0f)
							{
								fragments[fragIndex].color = diffusedTerm1 * col * light1.col + diffusedTerm2 * norm * light2.col;
							}

							else if(diffusedTerm1 > 0.0f)
							{
								fragments[fragIndex].color = diffusedTerm1 * col * light1.col;
							}
							else
							{
								fragments[fragIndex].color = diffusedTerm2 * col * light2.col;
							}
						}
					}
				}
			}
		}
	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;

    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));

    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));

    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */

void setPrimitiveBuffer(int vertCount)
{
	hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));
}

void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }

    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    setPrimitiveBuffer(vertCount);

    hipFree(dev_outVertex);
    hipMalloc((void**)&dev_outVertex, vertCount * sizeof(VertexOut));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */

void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);


    if(scene->run)
    {
        Triangle *dev_primitivesEnd;

        int numThreads = 256;
        int numBlocks;
        int numTriangles = vertCount/3;

    	Camera &cam = scene->cam;
    	Light &light1 = scene->light1;
    	Light &light2 = scene->light2;

    	//Do vertex shading
    	numBlocks = (vertCount + numThreads -1)/numThreads;
    	kernVertexShader<<<numBlocks, numThreads>>>(vertCount, width, height, dev_bufVertex, dev_outVertex, cam.cameraMatrix, cam.model);

    	//Do primitive assembly
    	numBlocks = (numTriangles + numThreads -1)/numThreads;
    	kernPrimitiveAssembly<<<numBlocks, numThreads>>>(numTriangles, dev_outVertex, dev_bufVertex, dev_primitives, dev_bufIdx, cam.dir);

    	//Back face culling
//    	dev_primitivesEnd = dev_primitives + numTriangles;
//    	dev_primitivesEnd = thrust::remove_if(thrust::device, dev_primitives, dev_primitivesEnd, keep());
//    	numTriangles = dev_primitivesEnd - dev_primitives;
////    	std::cout<<numTriangles;

    	//Clear the color and depth buffers
    	kernClearFragmentBuffer<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);

    	//Drawing axis
//    	kernDrawAxis<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);

    	//Rasterization per triangle
    	numBlocks = (numTriangles + numThreads -1)/numThreads;
    	kernRasterize<<<numBlocks, numThreads>>>(width, height, dev_depthbuffer, dev_primitives, numTriangles, cam, light1, light2);

    	scene->run = false;
    }

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);

    //Save image data to write to file
    hipMemcpy(scene->imageColor, dev_framebuffer, width*height*sizeof(glm::vec3), hipMemcpyDeviceToHost);

    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    hipFree(dev_outVertex);
    dev_outVertex = NULL;

    checkCUDAError("rasterizeFree");
}
