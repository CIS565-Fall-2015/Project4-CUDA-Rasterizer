#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <climits>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <glm/glm.hpp>
#include <glm/gtc/matrix_transform.hpp>

#include <util/checkCUDAError.h>
#include "rasterizeTools.h"
#include "sceneStructs.h"

/************************* Struct Definitions *********************************/

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
};
struct VertexOut {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;

    glm::vec3 worldPos;
    glm::vec3 worldNor;
};
struct Triangle {
    glm::vec3 pos[3];
    glm::vec3 nor[3];
    glm::vec3 col[3];

    glm::vec3 worldPos[3];
    glm::vec3 worldNor[3];
};
struct Fragment {
    glm::vec3 color;
    Triangle tri;
    glm::vec3 baryCoords;
    int z;
    bool valid;
};

static int width = 0;
static int height = 0;
static int bufIdxSize = 0;
static int vertCount = 0;

static int       *dev_bufIdx       = NULL;
static VertexIn  *dev_bufVertexIn  = NULL;
static VertexOut *dev_bufVertexOut = NULL;
static Triangle  *dev_primitives   = NULL;
static Fragment  *dev_depthbuffer  = NULL;
static glm::vec3 *dev_framebuffer  = NULL;

__device__ void printVec3(glm::vec3 v) {
    printf("(%f, %f, %f)\n", v.x, v.y, v.z);
}

/************************* Output to Screen ***********************************/

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__ void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

/**
 * Writes fragment colors to the framebuffer
 */
__global__ void render(int w, int h, Fragment *depthbuffer,
        glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/************************* Initialization *************************************/

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;

    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,    width * height * sizeof(Fragment));
    hipMemset( dev_depthbuffer, 0, width * height * sizeof(Fragment));

    hipFree(dev_bufVertexOut);
    hipMalloc(&dev_bufVertexOut,    width * height * sizeof(VertexOut));
    hipMemset( dev_bufVertexOut, 0, width * height * sizeof(VertexOut));

    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,    width * height * sizeof(glm::vec3));
    hipMemset( dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertexIn = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertexIn[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertexIn[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertexIn[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertexIn);
    hipMalloc(&dev_bufVertexIn, vertCount * sizeof(VertexIn));
    hipMemcpy( dev_bufVertexIn, bufVertexIn, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/************************* Rasterization Pipeline *****************************/

__global__ void clearDepthBuffer(int width, int height, Fragment *depthbuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    if (x < width && y < height) {
        int index = x + (y * width);

        depthbuffer[index].valid = false;
        depthbuffer[index].z = INT_MAX;
        depthbuffer[index].color = glm::vec3(.15, .15, .15);
    }
}

// Applies vertex transformations (from given model-view-projection matrix)
__global__ void vertexShader(int vertcount, VertexIn *verticesIn,
        VertexOut *verticesOut, glm::mat4 mvp) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < vertcount) {
        VertexIn vin = verticesIn[k];

        VertexOut vo;
        vo.pos = multiplyMV(mvp, glm::vec4(vin.pos, 1));
        vo.nor = multiplyMV(mvp, glm::vec4(vin.nor, 1));
        vo.col = vin.col;

        vo.worldPos = vin.pos;
        vo.worldNor = vin.nor;
        verticesOut[k] = vo;
    }
}

// Assembles sets of 3 vertices into Triangles.
__global__ void assemblePrimitives(int primitivecount, VertexOut *vertices,
        int *indices, Triangle *primitives) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < primitivecount) {
        VertexOut v[3];
        v[0] = vertices[indices[k*3  ]];
        v[1] = vertices[indices[k*3+1]];
        v[2] = vertices[indices[k*3+2]];

        Triangle tri;
        tri.pos[0] = v[0].pos;
        tri.pos[1] = v[1].pos;
        tri.pos[2] = v[2].pos;

        tri.nor[0] = v[0].nor;
        tri.nor[1] = v[1].nor;
        tri.nor[2] = v[2].nor;

        tri.col[0] = v[0].col;
        tri.col[1] = v[1].col;
        tri.col[2] = v[2].col;

        tri.worldPos[0] = v[0].worldPos;
        tri.worldPos[1] = v[1].worldPos;
        tri.worldPos[2] = v[2].worldPos;

        tri.worldNor[0] = v[0].worldNor;
        tri.worldNor[1] = v[1].worldNor;
        tri.worldNor[2] = v[2].worldNor;
        primitives[k] = tri;
    }
}

__device__ void storeFragment(float x, float y, float width, float height,
        Triangle tri, Fragment *fragments) {

    glm::vec3 bary = calculateBarycentricCoordinate(tri.pos, glm::vec2(x, y));
    glm::vec2 pos = fromNDC(x, y, width, height);
    int pixelIndex = pos.x + (pos.y * width);

    if (isBarycentricCoordInBounds(bary)) {
        Fragment prev = fragments[pixelIndex];

        float z = getZAtCoordinate(tri.worldPos, bary);
        int depth = z * INT_MAX;
        atomicMin(&fragments[pixelIndex].z, depth);

        if (fragments[pixelIndex].z == depth) {
            fragments[pixelIndex] = (Fragment) { glm::vec3(0, 0, 0), tri, bary, depth, true};
        }
    } else {
    }
}

// Scans across triangles to generate primitives (pixels).
__global__ void scanline(int width, int height, int tricount,
        Triangle *primitives, Fragment *fragments) {
    int k = (blockIdx.x * blockDim.x) + threadIdx.x;

    if (k < tricount) {
        Triangle tri = primitives[k];

        float ystep = 2.f / height;
        float xstep = 2.f / width;

        AABB bb = getAABBForTriangle(tri.pos);

        float ymin = glm::max(-1.f, (int) (bb.min.y / ystep) * ystep);
        float xmin = glm::max(-1.f, (int) (bb.min.x / xstep) * xstep);
        float ymax = glm::min(1.f, bb.max.y);
        float xmax = glm::min(1.f, bb.max.x);
        for (float y = ymin; y < ymax; y += ystep) {
            for (float x = xmin; x < xmax; x += xstep) {
                storeFragment(x, y, width, height, tri, fragments);
            }
        }
    }
}

__global__ void fragmentShader(int width, int height,
        Fragment *fragments, glm::vec3 light) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);

    if (x < width && y < height) {
        Fragment &frag = fragments[index];
        if (frag.valid) {
            glm::vec3 norm = barycentricInterpolate(frag.tri.worldNor, frag.baryCoords);
            frag.color = glm::abs(norm);
        } else {
        }
    }
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

    int tricount = bufIdxSize / 3;

    int sideLength1d = 16;
    dim3 blockSize1d(sideLength1d);
    dim3 vertBlockCount((vertCount + sideLength1d - 1) / sideLength1d);
    dim3 triBlockCount((tricount + sideLength1d - 1) / sideLength1d);

    Camera c;
    c.position = glm::vec3(0, 3, -10);
    c.view = glm::vec3(0, 0, 1);
    c.up = glm::vec3(0, -1, 0);
    c.light = glm::vec3(0, 4, 0);
    c.fovy = 45.f;

    glm::mat4 model = glm::mat4(1.f);
    glm::mat4 view = glm::lookAt(c.position, c.view, c.up);
    glm::mat4 persp = glm::perspective(c.fovy, 1.f, 1.f, 100.f);
    glm::mat4 mvp = persp * view * model;

    clearDepthBuffer<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);
    checkCUDAError("scan");

    vertexShader<<<vertBlockCount, blockSize1d>>>( vertCount,
            dev_bufVertexIn, dev_bufVertexOut, mvp);
    checkCUDAError("scan");

    // VertexOut -> Triangle
    assemblePrimitives<<<triBlockCount, blockSize1d>>>(tricount,
            dev_bufVertexOut, dev_bufIdx, dev_primitives);
    checkCUDAError("rasterize");

    // Triangle -> Fragment
    scanline<<<triBlockCount, blockSize1d>>>(width, height, tricount,
            dev_primitives, dev_depthbuffer);
    checkCUDAError("rasterize");

    // Fragment -> Fragment
    fragmentShader<<<blockCount2d, blockSize2d>>>(width, height,
            dev_depthbuffer, c.light);
    checkCUDAError("rasterize");

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertexOut);
    dev_bufVertexOut = NULL;

    hipFree(dev_bufVertexIn);
    dev_bufVertexIn = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
