#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include <glm/gtc/matrix_transform.hpp>
#include "rasterizeTools.h"
extern glm::vec3 camCoords;
glm::vec3 cam(0.0f, 3.0f, 3.0f);
#define MAX_DEPTH 1000000
struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
	glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
	float depth;
	int idepth;
	glm::vec3 nor;

};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static VertexOut *dev_bufTransformedVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static Fragment *dev_fragbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;



/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
	//printf("Width: %i, Height: %i", width, height);
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
	hipFree(dev_fragbuffer);
	hipMalloc(&dev_fragbuffer, width * height * sizeof(Fragment));
	hipMemset(dev_fragbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;
    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
	hipMalloc(&dev_bufTransformedVertex, vertCount * sizeof(VertexOut));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

__global__ void setDepth(Fragment* depth, int width) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);
	depth[index].depth = MAX_DEPTH;
	depth[index].idepth = MAX_DEPTH;
	depth[index].color = glm::vec3(0.0f, 0.0f, 0.0f);
}
 
 __device__ __host__ glm::vec4 mul(glm::mat4 m, glm::vec4 v) {
    return glm::vec4(m[0].x*v.x + m[1].x*v.y + m[2].x*v.z + m[3].x*v.w,
                 m[0].y*v.x + m[1].y*v.y + m[2].y*v.z + m[3].y*v.w,
                 m[0].z*v.x + m[1].z*v.y + m[2].z*v.z + m[3].z*v.w,
                 m[0].w*v.x + m[1].w*v.y + m[2].w*v.z + m[3].w*v.w);
 }

__global__ void vertexShader(VertexIn* inVerts, VertexOut* outVerts, int vertCount, glm::mat4 matrix) {
	int thrId = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thrId < vertCount) {
		//printf("%i oldVert: (%f, %f, %f) \n", thrId, inVerts[thrId].pos[0], inVerts[thrId].pos[1], inVerts[thrId].pos[2]);
		//printf("matrix: (%f, %f, %f, %f) \n", matrix[2][0], matrix[2][1], matrix[2][2], matrix[2][3]);
		//printf("matrix: (%f, %f, %f, %f) \n", matrix[3][0], matrix[3][1], matrix[3][2], matrix[3][3]);
		
		//glm::vec4 newVert = matrix * glm::vec4(inVerts[thrId].pos[0], inVerts[thrId].pos[1], inVerts[thrId].pos[2], 1.0f);
		//newVert[2] = (matrix[2][2] * inVerts[thrId].pos[2]) + matrix[2][3];
		//newVert[3] = (matrix[3][2] * inVerts[thrId].pos[2]) + matrix[3][3];
		glm::vec4 oldVert(inVerts[thrId].pos, 1.0f);
		glm::vec4 newVert = mul(matrix, oldVert);
		//printf("newVert 2: %f, new vert 3: %f \n", newVert[2], newVert[3]);
		if (newVert[3] != 0) {
			outVerts[thrId].pos = glm::vec3(newVert[0] / newVert[3], newVert[1] / newVert[3], newVert[2] / newVert[3]);
			outVerts[thrId].nor = inVerts[thrId].nor;
			outVerts[thrId].col = inVerts[thrId].col;
			//printf("%i newVert: (%f, %f, %f) \n", thrId, outVerts[thrId].pos[0], outVerts[thrId].pos[1], outVerts[thrId].pos[2]);
		}
		else {
			outVerts[thrId].pos = glm::vec3(newVert[0], newVert[1], newVert[2]);
			outVerts[thrId].nor = inVerts[thrId].nor;
			outVerts[thrId].col = inVerts[thrId].col;
			//printf("%i newVert: (%f, %f, %f) \n", thrId, outVerts[thrId].pos[0], outVerts[thrId].pos[1], outVerts[thrId].pos[2]);
		}
	}

}

__global__ void primitiveAssemble(VertexOut* verts, Triangle* tris, int vertCount) {
	int thrId = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thrId < vertCount) {
		if (thrId % 3 == 0) {
			tris[thrId/3].v[0] = verts[thrId];
			//printf("%i first: (%i, %i, %i) \n", thrId, verts[thrId].pos[0], verts[thrId].pos[1], verts[thrId].pos[2]);
		} 
		else if (thrId % 3 == 1) {
			tris[(thrId - 1)/3].v[1] = verts[thrId];
			//printf("%i second: (%i, %i, %i) \n", thrId, verts[thrId].pos[0], verts[thrId].pos[1], verts[thrId].pos[2]);
		}
		else {
			tris[(thrId - 2)/3].v[2] = verts[thrId];
			//printf("%i third: (%i, %i, %i) \n", thrId, verts[thrId].pos[0], verts[thrId].pos[1], verts[thrId].pos[2]);
		}
	}
}

__global__ void kernRasterize(Triangle* tris, Fragment* buf, int width, int height, int triCount) {
	int thrId = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (thrId < triCount) {

		glm::vec3 triangle[3];
		triangle[0] = tris[thrId].v[0].pos;
		triangle[1] = tris[thrId].v[1].pos;
		triangle[2] = tris[thrId].v[2].pos;
		//printf("1 (%f, %f, %f) \n", tris[thrId].v[0].pos[2], tris[thrId].v[1].pos[2], tris[thrId].v[2].pos[2]);
		/*printf("thrId: %i \n", thrId);
		printf("1 (%f, %f, %f) \n", tris[thrId].v[0].nor[0], tris[thrId].v[0].nor[1], tris[thrId].v[0].nor[2]);
		printf("2 (%f, %f, %f) \n", tris[thrId].v[1].nor[0], tris[thrId].v[1].nor[1], tris[thrId].v[1].nor[2]);
		printf("3 (%f, %f, %f) \n", tris[thrId].v[2].nor[0], tris[thrId].v[2].nor[1], tris[thrId].v[2].nor[2]);
		
		printf("normal (%f, %f, %f) \n", normal[0], normal[1], normal[2]);*/
		AABB bbox = getAABBForTriangle(triangle);
		//printf("min x: %f, max x: %f, min y: %f, max y: %f \n", bbox.min.x, bbox.max.x, bbox.min.y, bbox.max.y);
		float minX = (bbox.min.x + 1) * (width / 2.0f);
		float maxX = (bbox.max.x + 1) * (width / 2.0f);
		float minY = (bbox.min.y + 1) * (height / 2.0f);
		float maxY = (bbox.max.y + 1) * (height / 2.0f);
		for (int x = floor(minX); x < ceil(maxX); x++) {
			for (int y = floor(minY); y < ceil(maxY); y++) {
			
				float tempX = (((float)x / (float)width) * 2.0f) - 1.0f;
				float tempY = (((float)y / (float)height) * 2.0f) - 1.0f;
				//float minZ = (bbox.min.z + 1) * (width / 2.0f);
				//printf("z: %f, %f \n", minZ, bbox.min.z);
				
				//printf("depth int: %i \n", myDepth);
				if (tempX >= -1.0f && tempX <= 1.0f && tempY >= -1.0f && tempY <= 1.0f) {
					glm::vec3 baryCoord = calculateBarycentricCoordinate(triangle, glm::vec2(tempX, tempY));
					int myDepth = getZAtCoordinate(baryCoord, triangle) * 1000;
					atomicMin(&buf[x + y*width].idepth, myDepth);
					__syncthreads();
					//printf("buffer depth: %i \n", buf[x + y*width].idepth);
					if(isBarycentricCoordInBounds(baryCoord) && myDepth == buf[x + y*width].idepth) {
						//printf("index: %i   depth: %f   pixel: (%i, %i) \n", thrId, buf[x + y*width].depth, x, y);
						
						glm::vec3 normal = baryCoord[0] * tris[thrId].v[0].nor + baryCoord[1] * tris[thrId].v[1].nor + baryCoord[2] * tris[thrId].v[2].nor;
						buf[x + y*width].color = normal;
						buf[x + y*width].nor = normal;
						buf[x + y*width].depth = bbox.min.z;
					
					}
				}
				
			}
		}
	}
}

__global__ void fragmentShader(Fragment* depth, Fragment* frag, int width, int height) {
	glm::vec3 light(0.0f, 3.0f, 6.0f);
	

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * width);

	if (index < width * height) {
		if (depth[index].depth < MAX_DEPTH) {
			float diffuseTerm = glm::dot(glm::normalize(depth[index].nor), glm::normalize(-light));
			//printf("diff: %f norm: (%f, %f, %f) \n", diffuseTerm, depth[index].nor[0], depth[index].nor[1], depth[index].nor[2]);
			frag[index].color =  diffuseTerm * depth[index].color;//depth[index].nor; //glm::dot(light, depth[index].nor)*depth[index].color;
			//printf("(%f, %f, %f) \n", frag[index].color[0], frag[index].color[1], frag[index].color[2]);
		}
	}
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);
	dim3 blockSize1d(64);
	dim3 blockCount1d((vertCount + 64 - 1) / 64);
	//cam.x += camCoords.x;
	//cam.y += camCoords.y;
	//camCoords.x = 0.0;
	//camCoords.y = 0.0;
	glm::mat4 model = utilityCore::buildTransformationMatrix(glm::vec3(0.0f), glm::vec3(-camCoords.y, -camCoords.x, 0), glm::vec3(1.0f));
	glm::mat4 view = glm::lookAt(glm::vec3(0.0, 3, 3), glm::vec3(0, 0, 0), glm::vec3(0, 1, 0));
	glm::mat4 projection = glm::perspective<float>(50.0, (float)width / (float)height, 0.5f, 1000.0f);
	//glm::mat4 model = glm::mat4();
	glm::mat4 matrix = projection * view * model;

    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	//Set buffer to default value
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
	setDepth<<<blockCount2d, blockSize2d>>>(dev_depthbuffer, width);
	setDepth<<<blockCount2d, blockSize2d>>>(dev_fragbuffer, width);
	//Transfer from VertexIn to VertexOut (vertex shading)
	vertexShader<<<blockCount1d, blockSize1d>>>(dev_bufVertex, dev_bufTransformedVertex, vertCount, matrix);
	checkCUDAError("rasterize");
	int k;
	//std::cin >> k;
	//Transfer from VertexOut to Triangles (primitive assembly)
	primitiveAssemble<<<blockCount1d, blockSize1d>>>(dev_bufTransformedVertex, dev_primitives, vertCount);
	checkCUDAError("rasterize");
	
	//Scanline each triangle to get fragment color (rasterize)
	int triCount = vertCount / 3;
	blockCount1d = ((triCount + 64 - 1) / 64);
	//printf("tri count: %i, block count: %i \n", triCount, (triCount + 64 - 1) / 64);
	kernRasterize<<<blockCount1d, blockSize1d>>>(dev_primitives, dev_depthbuffer, width, height, triCount);
	checkCUDAError("rasterize");
    
    //Fragment shader
	fragmentShader<<<blockCount2d, blockSize2d>>>(dev_depthbuffer, dev_fragbuffer, width, height);
	checkCUDAError("Fragment Shader");

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_fragbuffer, dev_framebuffer);
    
	// Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
