#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
    // TODO
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
	float depth;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static VertexOut *dev_shadedVertices = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
	VertexOut *bufVertexOut = new VertexOut[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	hipFree(dev_shadedVertices);
	hipMalloc(&dev_shadedVertices, vertCount * sizeof(VertexOut));

    hipFree(dev_primitives);
	hipMalloc(&dev_primitives, bufIdxSize / 3 * sizeof(Triangle));
	hipMemset(dev_primitives, 0, bufIdxSize / 3 * sizeof(Triangle));

	delete bufVertex;
	delete bufVertexOut;

    checkCUDAError("rasterizeSetBuffers");
}

// minimal vertex shader
__global__
void minVertexShader(int vertCount, glm::mat4 tf, VertexIn *dev_verticesIn, VertexOut *dev_verticesOut) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < vertCount) {
		dev_verticesOut[i].pos = tfPoint(tf, dev_verticesIn[i].pos);
		dev_verticesOut[i].nor = tfDir(tf, dev_verticesIn[i].nor);
	}
}

// primitive assembly. 1D linear blocks expected
__global__
void primitiveAssembly(int numPrimitives, VertexOut *dev_vertices, Triangle *dev_primitives) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numPrimitives) {
		dev_primitives[i].v[0] = dev_vertices[i * 3];
		dev_primitives[i].v[1] = dev_vertices[i * 3 + 1];
		dev_primitives[i].v[2] = dev_vertices[i * 3 + 2];
	}
}

// scanline rasterization. 1D linear blocks expected
__global__
void scanlineRasterization(int w, int h, int numPrimitives, Triangle *dev_primitives, Fragment *dev_frags) {
	int i = (blockIdx.x * blockDim.x) + threadIdx.x;
	if (i < numPrimitives) {
		// get the AABB of the triangle
		glm::vec3 v[3];
		v[0] = dev_primitives[i].v[0].pos;
		v[1] = dev_primitives[i].v[1].pos;
		v[2] = dev_primitives[i].v[2].pos;
		printf("crash on allocating array?\n");
		AABB triangleBB = getAABBForTriangle(v);
		// triangle should have been "smooshed" to screen coordinates already.
		// walk and fill frags.
		float pixWidth = 1.0f / (float) w;
		float pixHeight = 1.0f / (float) h;

		for (int y = 0; y < h; y++) {
			for (int x = 0; x < w; x++) {
				// compute x y coordinates of the center of "this fragment"
				glm::vec2 fragCoord = glm::vec2(x * pixWidth + pixWidth * 0.5f,
					y * pixHeight + pixHeight * 0.5f);
				// check if it's in dev_primitives[i].v using bary
				glm::vec3 baryCoordinate = calculateBarycentricCoordinate(v, fragCoord);
				if (!isBarycentricCoordInBounds(baryCoordinate)) {
					continue;
				}
				// check depth using bary
				float zDepth = getZAtCoordinate(baryCoordinate, v);

				int fragIndex = x + (y * w);
				// if all things pass ok, then insert into fragment.
				if (zDepth < dev_frags[fragIndex].depth) {
					dev_frags[fragIndex].depth = zDepth;
					// interpolate color
					glm::vec3 interpColor = dev_primitives[i].v[0].col * baryCoordinate[0];
					interpColor += dev_primitives[i].v[1].col * baryCoordinate[1];
					interpColor += dev_primitives[i].v[2].col * baryCoordinate[2];
					dev_frags[fragIndex].color = interpColor;
					printf("crash on loading into frag buffer?\n");
				}
			}
		}
	}
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
	dim3 blockSize1d(sideLength2d * sideLength2d);

    dim3 blockCount2d_display((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	// 1) clear depth buffer with some default value. black seems reasonable.
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));

	// 2) vertex shade
	glm::mat4 ID = glm::mat4();
	dim3 blockCount1d_vertices((vertCount - 1) / blockSize1d.x + 1);

	minVertexShader <<<blockCount1d_vertices, blockSize1d>>>(vertCount, ID, dev_bufVertex, dev_shadedVertices);
	checkCUDAError("debug: vertex shading");

	// 3) primitive assembly
	int numPrimitives = bufIdxSize / 3;
	dim3 blockCount1d_primitives((numPrimitives - 1) / blockSize1d.x + 1);
	primitiveAssembly<<<blockCount1d_primitives, blockSize1d>>>(numPrimitives, dev_shadedVertices, dev_primitives);
	checkCUDAError("debug: primitive assembly");

	// 4) rasterization
	scanlineRasterization<<<blockCount1d_primitives, blockSize1d>>>(width, height, numPrimitives,
		dev_primitives, dev_depthbuffer);
	checkCUDAError("debug: scanline rasterization");

	// 5) fragment shading

	// 6) fragments to depth buffer

	// 7) depth buffer for storing depth testing fragments

	// 8) frag to frame buffer
    // Copy depthbuffer colors into framebuffer
	render << <blockCount2d_display, blockSize2d >> >(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
	sendImageToPBO << <blockCount2d_display, blockSize2d >> >(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

	hipFree(dev_shadedVertices);
	dev_shadedVertices = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
