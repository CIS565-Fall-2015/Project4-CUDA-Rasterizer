#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/remove.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"
#include <glm/gtc/matrix_transform.hpp>

extern glm::vec3 *imageColor;

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
};
struct Triangle {
    VertexOut vOut[3];
    VertexIn vIn[3];
    bool keep;
    glm::vec3 triNor;	//Used for back face culling
};
struct Fragment {
    glm::vec3 color;
    float depth;
};
struct Light {
	glm::vec3 pos;
	glm::vec3 col;
};

struct keep
{
	__host__ __device__ bool operator()(const Triangle t)
	{
		return (!t.keep);
	}
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;
static glm::mat4 matrix;
static glm::vec3 camDir;
static Light light;

//Things added
static VertexOut *dev_outVertex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

__global__
void kernVertexShader(int numVertices, int w, int h, VertexIn * inVertex, VertexOut *outVertex, glm::mat4 matrix)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numVertices)
	{
		glm::vec4 outPoint = glm::vec4(inVertex[index].pos.x, inVertex[index].pos.y, inVertex[index].pos.z, 1.0f);

		outPoint = matrix * outPoint;

		if(outPoint.w != 0)
			outPoint /= outPoint.w;

		//In NDC
//		outVertex[index].pos = glm::vec3(outPoint);

		//In Device Coordinates
		outVertex[index].pos.x = outPoint.x * w;
		outVertex[index].pos.y = outPoint.y * h;
		outVertex[index].pos.z = outPoint.z;

		outVertex[index].col = glm::vec3(0,0,1);
		outVertex[index].nor = inVertex[index].nor;

//		printf ("InVertex : %f %f \nOutVertex : %f %f \n\n", inVertex[index].pos.x, inVertex[index].pos.y, outVertex[index].pos.x, outVertex[index].pos.y);
	}
}

__global__
void kernPrimitiveAssembly(int numTriangles, VertexOut *outVertex, VertexIn *inVertex, Triangle *triangles, int* indices, glm::vec3 camDir)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		int k_3 = 3 * index;

		Triangle &t = triangles[index];
		glm::vec3 triNor = glm::normalize(inVertex[k_3].nor + inVertex[k_3+1].nor + inVertex[k_3+2].nor);

//		printf ("Tri Normal : %f %f %f\n", triNor.x, triNor.y, triNor.z);
//		printf ("Cam Dir : %f %f %f\n", camDir.x, camDir.y, camDir.z);

		if(glm::dot(triNor, camDir) > -0.0001f)
		{
			t.keep = false;
		}

		else
		{
			t.keep = true;

			t.vOut[0].pos = outVertex[indices[k_3]].pos;
			t.vOut[1].pos = outVertex[indices[k_3+1]].pos;
			t.vOut[2].pos = outVertex[indices[k_3+2]].pos;

			//TODO:  figure out the normals
			t.vOut[0].nor = outVertex[indices[k_3]].nor;
			t.vOut[1].nor = outVertex[indices[k_3+1]].nor;
			t.vOut[2].nor = outVertex[indices[k_3+2]].nor;

			t.triNor = triNor;
		}
	}
}

__global__
void kernDrawAxis(int w, int h, Fragment *fragments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h)
    {
		if((x - w*0.5f) == 0)
	    {
			fragments[index].color = glm::vec3(0, 1, 0);
	    }
		else if((y - h*0.5f) == 0)
		{
			fragments[index].color = glm::vec3(1, 0, 0);
		}
    }
}

__global__
void kernRasterizePerFragment()
{
	//Rasterization per Fragment
//	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//    int index = x + (y * w);
//
//    if (x < w && y < h)
//    {
//    	if((x - w*0.5f) == 0 || (y - h*0.5f) == 0)
//    	{
//    		fragments[index].color = glm::vec3(1);
//    	}
//    	else
//    	{
//			glm::vec2 point((x - w*0.5f), (y - h*0.5f));
//			for(int i=0; i<numTriangles; ++i)
//			{
//				glm::vec3 tri[3];
//				tri[0] = triangles[i].v[0].pos;
//				tri[1] = triangles[i].v[1].pos;
//				tri[2] = triangles[i].v[2].pos;
//
//	//    		AABB aabb = getAABBForTriangle(tri);
//				float signedArea = calculateSignedArea(tri);
//				glm::vec3 barycentric = calculateBarycentricCoordinate(tri, point);
//				if(isBarycentricCoordInBounds(barycentric))
//				{
//					fragments[index].color = glm::vec3(1);
//					fragments[index].depth = getZAtCoordinate(barycentric, tri);
//				}
//			}
//    	}
//    }
}

__global__
void kernRasterize(int w, int h, Fragment *fragments, Triangle *triangles, int numTriangles, glm::vec3 camDir, Light light)
{
	//Rasterization per triangle
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		Triangle &t = triangles[index];

		glm::vec3 tri[3];
		tri[0] = t.vOut[0].pos;
		tri[1] = t.vOut[1].pos;
		tri[2] = t.vOut[2].pos;

		AABB aabb = getAABBForTriangle(tri);
		glm::ivec3 min, max;

		//Attempted clipping
		min.x = glm::clamp(aabb.min.x, -(float)w*0.5f+1, (float)w*0.5f-1);
		min.y = glm::clamp(aabb.min.y, -(float)h*0.5f+1, (float)h*0.5f-1);
		max.x = glm::clamp(aabb.max.x, -(float)w*0.5f+1, (float)w*0.5f-1);
		max.y = glm::clamp(aabb.max.y, -(float)h*0.5f+1, (float)h*0.5f-1);

		for(int i=min.x-1; i<=max.x+1; ++i)
		{
			for(int j=min.y-1; j<=max.y+1; ++j)
			{
	//				printf("\nMax : %f %f %f\nMin : %f %f %f\n", aabb.max.x, aabb.max.y, aabb.max.z, aabb.min.x, aabb.min.y, aabb.min.z);
				glm::ivec2 point(i,j);
				glm::vec3 barycentric = calculateBarycentricCoordinate(tri, point);

				if(isBarycentricCoordInBounds(barycentric))
				{
					//Then fragment in the triangle.
					//Implement lambert shading

					//Interpolate normal
					glm::vec3 norm = barycentric.x * t.vOut[0].nor +
										barycentric.y * t.vOut[1].nor +
						                barycentric.z * t.vOut[2].nor;

//					glm::vec3 lightVector =
					fragments[int((i+w*0.5) + (j + h*0.5)*w)].color = glm::normalize(norm);//triangles[index].v[0].col;
//						fragments[int((i+w*0.5) + (j + h*0.5)*w)].color = t.triNor;
//						fragments[(x+1) * y].depth = getZAtCoordinate(barycentric, tri);
				}
			}
		}
	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }

    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    hipFree(dev_outVertex);
    hipMalloc((void**)&dev_outVertex, vertCount * sizeof(VertexOut));

    imageColor = new glm::vec3[width*height];

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
bool run = true;

void createCameraAndLight()
{
	//Camera stuff
	glm::vec3 camEye, camCenter, camUp;
	camEye = glm::vec3(0,0,5);
	camCenter = glm::vec3(0,0,0);
	camUp = glm::vec3(0,-1,0);

	glm::mat4 view = glm::lookAt(camEye, camCenter, camUp);
//	glm::mat4 projection = glm::frustum<float>(-1, 1, -1, 1, -1, 1);
	glm::mat4 projection = glm::perspective<float>(45.0f, float(width)/ float(height), -100.0f, 100.0f);
	glm::mat4 model = glm::mat4();
	glm::mat4 temp;

//	std::cout<<"View : "<<std::endl;
//	utilityCore::printMat4(view);
//	std::cout<<std::endl<<"Projection : "<<std::endl;
//	utilityCore::printMat4(projection);
//	std::cout<<std::endl<<"Model : "<<std::endl;
//	utilityCore::printMat4(model);
//	std::cout<<std::endl;

	matrix = projection * view * model;
	camDir = glm::normalize(camCenter - camEye);
	light.pos = glm::vec3(1,1,1);
	light.col = glm::vec3(1,1,1);
}

void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);


    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

    int numThreads = 256;
    int numBlocks;
    int numTriangles = vertCount/3;

    Triangle *dev_primitivesEnd;

    if(run)
    {
    	createCameraAndLight();

    	numBlocks = (vertCount + numThreads -1)/numThreads;
    	kernVertexShader<<<numBlocks, numThreads>>>(vertCount, width, height, dev_bufVertex, dev_outVertex, matrix);

    	numBlocks = (numTriangles + numThreads -1)/numThreads;
    	kernPrimitiveAssembly<<<numBlocks, numThreads>>>(numTriangles, dev_outVertex, dev_bufVertex, dev_primitives, dev_bufIdx, camDir);

    	std::cout<<"Num Triangles before : "<<numTriangles<<std::endl;
    	dev_primitivesEnd = dev_primitives + numTriangles;
    	dev_primitivesEnd = thrust::remove_if(thrust::device, dev_primitives, dev_primitivesEnd, keep());
    	numTriangles = dev_primitivesEnd - dev_primitives;
    	std::cout<<"Num Triangles after : "<<numTriangles<<std::endl;

    	kernDrawAxis<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);

    	numBlocks = (numTriangles + numThreads -1)/numThreads;
    	kernRasterize<<<numBlocks, numThreads>>>(width, height, dev_depthbuffer, dev_primitives, numTriangles, camDir, light);

//    	numBlocks = (width*height + numThreads -1)/numThreads;
//    	kernRasterize<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_primitives, numTriangles);

    	run = false;
    }

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);

    hipMemcpy(imageColor, dev_framebuffer, width*height*sizeof(glm::vec3), hipMemcpyDeviceToHost);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    hipFree(dev_outVertex);
    dev_outVertex = NULL;

    checkCUDAError("rasterizeFree");
}
