#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"
#include <glm/gtc/matrix_transform.hpp>

extern glm::vec3 *imageColor;

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
	glm::vec3 pos;
	glm::vec3 nor;
	glm::vec3 col;
};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
    float depth;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;
static glm::mat4 matrix;

//Things added
static VertexOut *dev_outVertex = NULL;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

__global__
void kernVertexShader(int numVertices, int w, int h, VertexIn * inVertex, VertexOut *outVertex, glm::mat4 matrix)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numVertices)
	{
		glm::vec4 outPoint = glm::vec4(inVertex[index].pos.x, inVertex[index].pos.y, inVertex[index].pos.z, 1.0f);

		outPoint = matrix * outPoint;

		if(outPoint.w != 0)
			outPoint /= outPoint.w;

		//In NDC
//		outVertex[index].pos = glm::vec3(outPoint);

		//In Device Coordinates
		outVertex[index].pos.x = outPoint.x * w;
		outVertex[index].pos.y = outPoint.y * h;
		outVertex[index].pos.z = outPoint.z;


//		printf ("InVertex : %f %f \nOutVertex : %f %f \n\n", inVertex[index].pos.x, inVertex[index].pos.y, outVertex[index].pos.x, outVertex[index].pos.y);
	}
}

__global__
void kernPrimitiveAssembly(int numTriangles, VertexOut *outVertex, Triangle *triangles, int* indices)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		int k_3 = 3 * index;
		triangles[index].v[0] = outVertex[indices[k_3]];
		triangles[index].v[1] = outVertex[indices[k_3+1]];
		triangles[index].v[2] = outVertex[indices[k_3+2]];

//		printf ("Triangle : %d\n", index);
//		printf ("Vertex 1 : %f %f\n", triangles[index].v[0].pos.x, triangles[index].v[0].pos.y);
//		printf ("Vertex 2 : %f %f\n", triangles[index].v[1].pos.x, triangles[index].v[1].pos.y);
//		printf ("Vertex 3 : %f %f\n", triangles[index].v[2].pos.x, triangles[index].v[2].pos.y);
	}
}

__global__
void kernDrawAxis(int w, int h, Fragment *fragments)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h)
    {
		if((x - w*0.5f) == 0 || (y - h*0.5f) == 0)
	    {
			fragments[index].color = glm::vec3(1);
	    }
    }
}

__global__
void kernRasterize(int w, int h, Fragment *fragments, Triangle *triangles, int numTriangles)
{
	//Rasterization per Fragment
//	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//    int index = x + (y * w);
//
//    if (x < w && y < h)
//    {
//    	if((x - w*0.5f) == 0 || (y - h*0.5f) == 0)
//    	{
//    		fragments[index].color = glm::vec3(1);
//    	}
//    	else
//    	{
//			glm::vec2 point((x - w*0.5f), (y - h*0.5f));
//			for(int i=0; i<numTriangles; ++i)
//			{
//				glm::vec3 tri[3];
//				tri[0] = triangles[i].v[0].pos;
//				tri[1] = triangles[i].v[1].pos;
//				tri[2] = triangles[i].v[2].pos;
//
//	//    		AABB aabb = getAABBForTriangle(tri);
//				float signedArea = calculateSignedArea(tri);
//				glm::vec3 barycentric = calculateBarycentricCoordinate(tri, point);
//				if(isBarycentricCoordInBounds(barycentric))
//				{
//					fragments[index].color = glm::vec3(1);
//					fragments[index].depth = getZAtCoordinate(barycentric, tri);
//				}
//			}
//    	}
//    }

	//Rasterization per triangle
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < numTriangles)
	{
		glm::vec3 tri[3];
		tri[0] = triangles[index].v[0].pos;
		tri[1] = triangles[index].v[1].pos;
		tri[2] = triangles[index].v[2].pos;

		AABB aabb = getAABBForTriangle(tri);
		for(int i=aabb.min.x-1; i<aabb.max.x+1; ++i)
		{
			for(int j=aabb.min.y-1; j<aabb.max.y+1; ++j)
			{
//				printf("\nMax : %f %f %f\nMin : %f %f %f\n", aabb.max.x, aabb.max.y, aabb.max.z, aabb.min.x, aabb.min.y, aabb.min.z);
				glm::ivec2 point(i,j);
//				//		float signedArea = calculateSignedArea(tri);
				glm::vec3 barycentric = calculateBarycentricCoordinate(tri, point);
				if(isBarycentricCoordInBounds(barycentric))
				{
					fragments[int((i+w*0.5) + (j + h*0.5)*w)].color = glm::vec3(1);
////					fragments[(x+1) * y].depth = getZAtCoordinate(barycentric, tri);
				}
			}
		}
	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));

    hipFree(dev_outVertex);
    hipMalloc((void**)&dev_outVertex, vertCount * sizeof(VertexOut));

    imageColor = new glm::vec3[width*height];

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
bool run = true;

void createCamera()
{
	//Camera stuff
	glm::vec3 camEye, camCenter, camUp;
	camEye = glm::vec3(0,0,-5);
	camCenter = glm::vec3(0,0,0);
	camUp = glm::vec3(0,-1,0);

	glm::mat4 view = glm::lookAt(camEye, camCenter, camUp);
//	glm::mat4 projection = glm::frustum<float>(-1, 1, -1, 1, -1, 1);
	glm::mat4 projection = glm::perspective<float>(45.0f, float(width)/ float(height), -100.0f, 100.0f);
	glm::mat4 model = glm::mat4();
	glm::mat4 temp;

//	std::cout<<"View : "<<std::endl;
//	utilityCore::printMat4(view);
//	std::cout<<std::endl<<"Projection : "<<std::endl;
//	utilityCore::printMat4(projection);
//	std::cout<<std::endl<<"Model : "<<std::endl;
//	utilityCore::printMat4(model);
//	std::cout<<std::endl;

	matrix = projection * view * model;
}

void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);


    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

    int numThreads = 256;
    int numBlocks;
    int numTriangles = vertCount/3;

    if(run)
    {
    	createCamera();

    	numBlocks = (vertCount + numThreads -1)/numThreads;
    	kernVertexShader<<<numBlocks, numThreads>>>(vertCount, width, height, dev_bufVertex, dev_outVertex, matrix);

    	numBlocks = (numTriangles + numThreads -1)/numThreads;
    	kernPrimitiveAssembly<<<numBlocks, numThreads>>>(numTriangles, dev_outVertex, dev_primitives, dev_bufIdx);

    	kernDrawAxis<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer);
//    	numBlocks = (width*height + numThreads -1)/numThreads;
    	kernRasterize<<<numBlocks, numThreads>>>(width, height, dev_depthbuffer, dev_primitives, numTriangles);
//    	kernRasterize<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_primitives, numTriangles);

    	run = false;
    }

    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);

    hipMemcpy(imageColor, dev_framebuffer, width*height*sizeof(glm::vec3), hipMemcpyDeviceToHost);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    hipFree(dev_outVertex);
    dev_outVertex = NULL;

    checkCUDAError("rasterizeFree");
}
