#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>
#include "rasterizeTools.h"

struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};
struct VertexOut {
    // TODO
	glm::vec3 pos;
	glm::vec3 nor;
    glm::vec3 col;

};
struct Triangle {
    VertexOut v[3];
};
struct Fragment {
    glm::vec3 color;
	glm::vec3 nor;
	float z;	
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static VertexOut *dev_bufVertex_out = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}

// Writes fragment colors to the framebuffer
__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        framebuffer[index] = depthbuffer[index].color;
    }
}

//vertex shader function
__global__
void kern_vertex_shader(VertexIn *dev_bufVertex_in, VertexOut *dev_bufVertex_out, int vertCount,glm::mat4 trans,glm::mat4 trans_inv_T) //trans = proj*view*model
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	//simple version for doing nothing

	
	if( index < vertCount)
	{
		VertexIn cur_v_in = dev_bufVertex_in[index];
		//calculate pos 
		dev_bufVertex_out[index].pos = glm::vec3(trans*glm::vec4(cu_v_in.pos,1.f));
		//calculate normal
		dev_bufVertex_out[index].nor = glm::vec3(trans_inv_T*glm::vec4(cu_v_in.nor,1.f));
		//calculate color
		dev_bufVertex_out[index].col = cur_v_in[index].col;
	}

}


//primitives assembly
__global__ 
void kern_premitive_assemble(VertexOut* dev_bufVertex_out,int* dev_bufIdx,Triangle* dev_primitives,int num_of_primitives)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index<num_of_primitives)
	{
		int v1_index = dev_bufIdx[3*index+0];
		int v2_index = dev_bufIdx[3*index+1];
		int v3_index = dev_bufIdx[3*index+2];

		dev_primitives[index].v[0] = dev_bufVertex_out[v1_index];
		dev_primitives[index].v[1] = dev_bufVertex_out[v2_index];
		dev_primitives[index].v[2] = dev_bufVertex_out[v3_index];


	}
}

//Rasterization
__global__ 
void kern_rasterization(Triangle* dev_primitives,Fragment *dev_depthbuffer, int num_of_primitives, int width, int height)
{
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if( index < num_of_primitives)
	{
		Triangle cur_triangle = dev_primitives[index];
		glm::vec3 m_tri[3] = {cur_triangle.v[0].pos,cur_triangle.v[1].pos,cur_triangle.v[2].pos};
		glm::vec3 m_normals[3] = {cur_triangle.v[0].nor,cur_triangle.v[1].nor,cur_triangle.v[2].nor};
		glm::vec3 m_colors[3] = {cur_triangle.v[0].col,cur_triangle.v[1].col,cur_triangle.v[2].col};
		AABB cur_AABB = getAABBForTriangle(m_tri);

		float min_x = max(cur_AABB.min.x,-1) ;
		float min_y = max(cur_AABB.min.y,-1);
		float max_x = min(cur_AABB.max.x,1);
		float max_y = min(cur_AABB.max.y,1);

		float dx = 2.f/width;
		float dy = 2.f/height;

		int min_x_idx = max((min_x+1)/dx,0);
		int min_y_idx = max((min_y+1)/dy,0);
		int max_x_idx = min((max_x+1)/dx,width-1);
		int max_y_idx = min((max_y+1)/dy,height-1);


		
		//first try the center sampling method
		
		for(int i = min_y_idx;i<=max_y_idx;i++)
		{
			for(int j = min_x_idx ; j<=max_y_idx ;j++)
			{
				int buffer_index = i*width + j;

				float cur_y = ((float)i*2+1.f)/(float)height;
				float cur_x = ((float)j*2+1.f)/(float)width;

				glm::vec2 cur_vec2 (cur_x,cur_y);

				glm::vec3 b_c = calculateBarycentricCoordinate(m_tri,cur_vec2);
				bool is_inside = isBarycentricCoordInBounds(b_c);

				if(is_inside)
				{
					float cur_z = getZAtCoordinate(b_c,m_tri);
					if(cur_z <= 1 && cur_z >= -1) //within the range
					{
						if(dev_depthbuffer[buffer_index].z<cur_z)
						{
							dev_depthbuffer[buffer_index].z = cur_z;
							
							//interpolate the color
							
							dev_depthbuffer[buffer_index].col =m_colors[0]*b_c.x +m_colors[1]*b_c.y+m_colors[2]*b_c.z;
							
							//interpolate the normal
							dev_depthbuffer[buffer_index].nor = m_normals[0]*b_c.x +m_normals[1]*b_c.y+m_normals[2]*b_c.z;
						}
					}
				}

			}
		}






	}
	
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));
    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	hipFree(dev_bufVertex_out);
    hipMalloc(&dev_bufVertex_out, vertCount * sizeof(VertexOut));

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, bufIdxSize / 3 * sizeof(Triangle));
    //hipMemset(dev_primitives, 0, bufIdxSize / 3 * sizeof(Triangle));

    checkCUDAError("rasterizeSetBuffers");
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);
	
    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	//vertex shader 
	dim3 blockSize1d (THREADS_PER_BLOCK);
	dim3 blockCount1d (vertCount/THREADS_PER_BLOCK+1);

	kern_vertex_shader<<<blockCount1d,blockSize1d>>>(dev_bufVertex, dev_bufVertex_out, vertCount, glm::mat4(1.f), glm::mat4(1.f));

	//primitive assembler
	int num_of_primitives = bufIdxSize/3;
	blockCount1d.x = num_of_primitives/THREADS_PER_BLOCK+1;

	
	kern_premitive_assemble(dev_bufVertex_out,dev_bufIdx,dev_primitives, num_of_primitives);

	//rasterization


    // Copy depthbuffer colors into framebuffer
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer);
    // Copy framebuffer into OpenGL buffer for OpenGL previewing
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

	hipFree(dev_bufVertex_out);
    dev_bufVertex_out = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;

    checkCUDAError("rasterizeFree");
}
