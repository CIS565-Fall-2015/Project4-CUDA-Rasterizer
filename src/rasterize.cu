#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include "rasterize.h"

#include <cstdint>
#include <limits>
#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <thrust/random.h>
#include <util/checkCUDAError.h>


#include "rasterizeTools.h"

#include "glm/gtc/matrix_transform.hpp"
//#include "glm/gtx/component_wise.hpp"

#define BACKGROUND_COLOR (glm::vec3(0.0f))


struct VertexIn {
    glm::vec3 pos;
    glm::vec3 nor;
    glm::vec3 col;
    // TODO (optional) add other vertex attributes (e.g. texture coordinates)
};

 
struct VertexOut {
    // TODO
	glm::vec4 pos;	//in NDS
	glm::vec3 color;

	glm::vec3 pos_eye_space;
	glm::vec3 noraml_eye_space;
	
	float divide_w_clip;

	glm::vec2 uv;
};

struct Edge
{
	VertexOut v[2];

	float x, z;
	float dx, dz;


	//
	//VertexOut cur_v;	//used for interpolate between a scan line
	float gap_y;
};




//struct FragmentIn
//{
//	bool shade;
//
//	glm::vec3 color;
//	glm::vec3 normal_eye_space;
//	glm::vec2 uv;
//
//	float depth;
//
//	
//	//__host__ __device__ FragmentIn(){ shade = false; depth = FLT_MAX; }
//};







struct Triangle {
    VertexOut v[3];
};
struct Fragment {
	//bool shade;
	bool has_fragment;

	glm::vec3 color;
	glm::vec3 normal_eye_space;
	glm::vec2 uv;

	glm::vec3 pos_eye_space;
	//float depth;

    //glm::vec3 color;
};

static int width = 0;
static int height = 0;
static int *dev_bufIdx = NULL;
static VertexIn *dev_bufVertex = NULL;
static Triangle *dev_primitives = NULL;
static Fragment *dev_depthbuffer = NULL;
static glm::vec3 *dev_framebuffer = NULL;
static int bufIdxSize = 0;
static int vertCount = 0;

static int triCount = 0;

//static FragmentIn * dev_fragments = NULL;
static int * dev_depth = NULL;

static ShaderMode shaderMode = SHADER_NORMAL;
static Light hst_lights[NUM_LIGHTS] = {
	Light{ DIRECTION_LIGHT
	, glm::vec3(0.5f, 0.5f, 0.5f)
	, glm::normalize(glm::vec3(1.0f, -2.0f, 1.0f)), false },
	Light{ DIRECTION_LIGHT
	,  glm::vec3(0.5f, 0.5f, 0.5f)
	, glm::normalize(glm::vec3(0.0f, 0.0f, -1.0f)), true } };

static int lightsCount = NUM_LIGHTS;
static Light* dev_lights = NULL;



void changeShaderMode()
{
	shaderMode = (ShaderMode)((shaderMode+1)%2);
}


/**
 * Kernel that writes the image to the OpenGL PBO directly.
 */
__global__
void sendImageToPBO(uchar4 *pbo, int w, int h, glm::vec3 *image) {
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * w);

    if (x < w && y < h) {
        glm::vec3 color;
        color.x = glm::clamp(image[index].x, 0.0f, 1.0f) * 255.0;
        color.y = glm::clamp(image[index].y, 0.0f, 1.0f) * 255.0;
        color.z = glm::clamp(image[index].z, 0.0f, 1.0f) * 255.0;
        // Each thread writes one pixel location in the texture (textel)
        pbo[index].w = 0;
        pbo[index].x = color.x;
        pbo[index].y = color.y;
        pbo[index].z = color.z;
    }
}



/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h) {
    width = w;
    height = h;
    hipFree(dev_depthbuffer);
    hipMalloc(&dev_depthbuffer,   width * height * sizeof(Fragment));
    hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    
	//hipFree(dev_fragments);
	//hipMalloc(&dev_fragments, width * height *sizeof(FragmentIn));
	//hipMemset(dev_fragments, 0, width * height * sizeof(FragmentIn));
	
	hipFree(dev_depth);
	hipMalloc(&dev_depth, width * height *sizeof(int));
	//hipMemset(dev_depth, INT_MAX, width * height * sizeof(int));
	
	hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
    hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));





    checkCUDAError("rasterizeInit");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

	//MY
	triCount = vertCount / 3;
	/////////

    hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

    hipFree(dev_primitives);
    hipMalloc(&dev_primitives, vertCount / 3 * sizeof(Triangle));
    hipMemset(dev_primitives, 0, vertCount / 3 * sizeof(Triangle));






	////init lights
	hipFree(dev_lights);
	hipMalloc(&dev_lights, lightsCount*sizeof(Light));
	hipMemcpy(dev_lights, hst_lights, lightsCount * sizeof(Light), hipMemcpyHostToDevice);
	////////////


    checkCUDAError("rasterizeSetBuffers");
}






//-------------------------------------------------------------------------------
// Vertex Shader
//-------------------------------------------------------------------------------

/**
* each thread copy info for one vertex
*/
__global__ 
void kernVertexShader(int N,glm::mat4 M, glm::mat4 M_model_view, glm::mat4 M_normal_view, VertexIn * dev_vertex, Triangle * dev_triangles)
{
	int vertexId = blockDim.x * blockIdx.x + threadIdx.x;
	

	if (vertexId < N)
	{
		int triangleId = vertexId / 3;
		int i = vertexId - triangleId * 3;
		VertexIn & vi = dev_vertex[vertexId];
		VertexOut & vo = dev_triangles[triangleId].v[i];

		vo.pos = M * glm::vec4(vi.pos, 1);
		
		//printf("%f,%f\n", vo.pos.x, vo.pos.w);
		
		vo.noraml_eye_space = glm::vec3(M_normal_view * glm::vec4(vi.nor, 0));
		vo.pos_eye_space = glm::vec3(M_model_view * glm::vec4(vi.pos, 1));


		vo.color = vi.col;

		//TODO: UV etc...
		
	}
}

/**
* MY:
* 
* VertexIn dev_bufVertex => Triangle VertexOut
* M model-view
*/
void vertexShader(const glm::mat4 & M, const glm::mat4 & M_model_view, const glm::mat4 & inv_trans_M)
{
	const int blockSize = 192;
	dim3 blockCount( (vertCount + blockSize - 1 )/blockSize );

	// get M, M_normal_view

	kernVertexShader << <blockCount, blockSize >> >(vertCount, M, M_model_view, inv_trans_M, dev_bufVertex, dev_primitives);
}

//------------------------------------------------------------------------------








//MY
//-------------------------------------------------------------------------------
// Rasterization
//-------------------------------------------------------------------------------

__host__ __device__
VertexOut interpolateVertexOut(const VertexOut & a, const VertexOut & b,float u)
{
	VertexOut c;

	if (u < 0.0f){ u = 0.0f; }
	else if (u > 1.0f){ u = 1.0f; }

	c.divide_w_clip = (1.0f - u) * a.divide_w_clip + u * b.divide_w_clip;
	
	c.pos = (1.0f - u) * a.pos + u * b.pos;
	c.color = (1.0f - u) * a.color + u * b.color;
	c.uv = (1.0f - u) * a.uv + u * b.uv;

	c.pos_eye_space = (1.0f - u) * a.pos_eye_space + u * b.pos_eye_space;
	c.noraml_eye_space = glm::normalize((1.0f - u) * a.noraml_eye_space + u * b.noraml_eye_space);
	
	return c;
}


//e.v[0] is the one with smaller y value
//scan from v[0] to v[1]
__device__ 
void constructEdge(Edge & e, const VertexOut & v0, const VertexOut & v1)
{
	if (v0.pos.y <= v1.pos.y)
	{
		e.v[0] = v0;
		e.v[1] = v1;
	}
	else
	{
		e.v[0] = v1;
		e.v[1] = v0;
	}

	//TODO: other members
	//e.cur_v = e.v[0];
	e.gap_y = 0.0f;

}


__device__
float initEdge(Edge & e, float y)
{
	e.gap_y = e.v[1].pos.y - e.v[0].pos.y;
	
	e.dx = (e.v[1].pos.x - e.v[0].pos.x) / e.gap_y;
	e.dz = (e.v[1].pos.z - e.v[0].pos.z) / e.gap_y;
	e.x = e.v[0].pos.x + (y - e.v[0].pos.y) * e.dx;
	e.z = e.v[0].pos.z + (y - e.v[0].pos.y) * e.dz;

	//if (e.x < 0)
	//{
	//	printf("%f,%f \n", e.x, e.dx);
	//}

	return (y - e.v[0].pos.y) / e.gap_y;
}

__device__
void updateEdge(Edge & e)
{
	e.x += e.dx;
	e.z += e.dz;
}



__device__
void drawOneScanLine(int width, const Edge & e1, const Edge & e2, int y,float u1,float u2, Fragment * fragments, int * depth)
{
	// Find the starting and ending x coordinates and
	// clamp them to be within the visible region
	int x_left = (int)(ceilf(e1.x) + EPSILON);
	int x_right = (int)(ceilf(e2.x) + EPSILON);

	

	if (x_left < 0)
	{
		x_left = 0;
	}
	
	if (x_right >= width)
	{
		x_right = width-1;
	}

	// Discard scanline with no actual rasterization and also
	// ensure that the length is larger than zero
	if (x_left >= x_right) { return; }


	//TODO: get two interpolated segment end points
	VertexOut cur_v_e1 = interpolateVertexOut(e1.v[0], e1.v[1], u1);
	VertexOut cur_v_e2 = interpolateVertexOut(e2.v[0], e2.v[1], u2);


	//Initialize attributes
	float dz = (e2.z - e1.z) / (e2.x - e1.x);
	float z = e1.z + (x_left - e1.x) * dz;


	//Interpolate
	//printf("%d,%d\n", x_left, x_right);
	float gap_x = x_right - x_left;
	for (int x = x_left; x < x_right; ++x)
	{
		int idx = x + y * width;



		// Z-buffer comparision
		VertexOut p = interpolateVertexOut(cur_v_e1, cur_v_e2, ((float)(x-x_left)) / gap_x);
		



		////atomic 
		//int assumed;
		//int* address = &depth[idx];
		//int old = *address;
		////lock method, don't know why it doesn't work on some cases
		//do{
		//	assumed = old;
		//	old = atomicCAS(address, assumed, 1);
		//} while (assumed != old);
		//if (*address == 0)
		//{
		//	printf(" -%d- ", *address);
		//}
		
		int z_int = (int)(z * INT_MAX);
		int* address = &depth[idx];

		atomicMin(address, z_int);

		//if (fragments[idx].shade == false)
		//{
		//	fragments[idx].shade = true;
		//	fragments[idx].depth = FLT_MAX;
		//}


		//if (z < fragments[idx].depth)
		if (*address == z_int)
		{
			//fragments[idx].depth = z;
			fragments[idx].color = p.color / p.divide_w_clip;
			fragments[idx].normal_eye_space = glm::normalize( p.noraml_eye_space / p.divide_w_clip );
			fragments[idx].pos_eye_space = p.noraml_eye_space / p.divide_w_clip;
			fragments[idx].uv = p.uv / p.divide_w_clip;

			fragments[idx].has_fragment = true;
		}
		

		////unlock
		//old = *address;
		//do{
		//	assumed = old;
		//	old = atomicCAS(address, assumed, 0);
		//} while (assumed != old);
		////if (*address == 1)
		////{
		////	printf("%d,%d\t", *address,old);
		////}
		

		z += dz;
	}
}








/**
* Rasterize the area between two edges as the left and right limit.
* e1 - longest y span
*/
__device__
void drawAllScanLines(int width, int height, Edge  e1, Edge  e2, Fragment * fragments, int * depth)
{
	// Discard horizontal edge as there is nothing to rasterize
	if (e2.v[1].pos.y - e2.v[0].pos.y == 0.0f) { return; }

	// Find the starting and ending y positions and
	// clamp them to be within the visible region
	int y_bot = (int)(ceilf(e2.v[0].pos.y) + EPSILON);
	int y_top = (int)(ceilf(e2.v[1].pos.y) + EPSILON);

	if (y_bot < 0)
	{
		y_bot = 0;
	}

	if (y_top >= height)
	{
		y_top = height-1;
	}


	//Initialize edge's structure
	float u1_base = initEdge(e1, (float)y_bot);
	initEdge(e2, (float)y_bot);




	for (int y = y_bot; y < y_top; ++y)
	{
		float u2 = ((float)(y - y_bot)) / e2.gap_y;
		float u1 = u1_base + ((float)(y - y_bot)) / e1.gap_y;
		if (e1.x <= e2.x)
		{
			drawOneScanLine(width, e1, e2, y ,u1,u2, fragments, depth);
		}
		else
		{
			drawOneScanLine(width, e2, e1, y, u2,u1, fragments, depth);
		}

		//update edge
		updateEdge(e1);
		updateEdge(e2);
	}
}





/**
* Each thread handles one triangle
* rasterization
*/
__global__
void kernScanLineForOneTriangle(int width,int height
, Triangle * triangles, Fragment * depth_fragment, int * depth)
{
	int triangleId = blockDim.x * blockIdx.x + threadIdx.x;

	Triangle tri = triangles[triangleId];	//copy

	//currently tri.v are in clipped coordinates
	//need to transform to viewport coordinate
	for (int i = 0; i < 3; i++)
	{
		if (tri.v[i].pos.w == 0.0f)
		{
			tri.v[i].divide_w_clip = 0;
		}
		else
		{
			tri.v[i].divide_w_clip = 1.0f / tri.v[i].pos.w;
		}
		
		
		
		//view port
		tri.v[i].pos.x = 0.5f * (float)width * (tri.v[i].pos.x * tri.v[i].divide_w_clip + 1.0f);
		tri.v[i].pos.y = 0.5f * (float)height * (tri.v[i].pos.y * tri.v[i].divide_w_clip + 1.0f);
		tri.v[i].pos.z = 0.5f * (tri.v[i].pos.z * tri.v[i].divide_w_clip + 1.0f);
		tri.v[i].pos.w = 1.0f;

		////1.#QNANO
		//if (tri.v[i].divide_w_clip == 0.0f)
		//{
		//	//printf("%f,%f\n", tri.v[i].pos.x, tri.v[i].divide_w_clip);
		//}
		

		//perspective correct interpolation
		tri.v[i].color *= tri.v[i].divide_w_clip;
		tri.v[i].noraml_eye_space *= tri.v[i].divide_w_clip;
		tri.v[i].pos_eye_space *= tri.v[i].divide_w_clip;
		tri.v[i].uv *= tri.v[i].divide_w_clip;
		


		////////

	}


	//build edge
	// for line scan
	Edge edges[3];

	constructEdge(edges[0], tri.v[0], tri.v[1]);
	constructEdge(edges[1], tri.v[1], tri.v[2]);
	constructEdge(edges[2], tri.v[2], tri.v[0]);

	//if (!(edges[0].x >= 0.0f))
	//{
	//	printf("%f,%f\n", edges[0].x, edges[0].dx);
	//}
	//if (!(edges[1].x >= 0.0f))
	//{
	//	printf("%f,%f\n", edges[1].x, edges[1].dx);
	//}
	//if (!(edges[2].x >= 0.0f))
	//{
	//	printf("%f,%f\n", edges[2].x, edges[2].dx);
	//}


	//Find the edge with longest y span
	float maxLength = 0.0f;
	int longEdge = -1;
	for (int i = 0; i < 3; ++i)
	{
		float length = edges[i].v[1].pos.y - edges[i].v[0].pos.y;
		if (length > maxLength)
		{
			maxLength = length;
			longEdge = i;
		}
	}


	// get indices for other two shorter edges
	int shortEdge0 = (longEdge + 1) % 3;
	int shortEdge1 = (longEdge + 2) % 3;

	// Rasterize two parts separately
	drawAllScanLines(width, height, edges[longEdge], edges[shortEdge0], depth_fragment, depth);
	drawAllScanLines(width, height, edges[longEdge], edges[shortEdge1], depth_fragment, depth);

	

}

//---------------------------------------------------------------------------



//-------------------------------------------------------------------------------
// Fragment Shader
//-------------------------------------------------------------------------------

//__global__ 
//void fragmentShader(int width, int height, Fragment* depthBuffer, FragmentIn* fragments   )
//{
//	//currently
//	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
//	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
//
//	if (x < width && y < height)
//	{
//		int index = x + y*width;
//
//		if (fragments[index].shade)
//		{
//			//depthBuffer[index].color = glm::vec3(1.0f);
//
//			//test: normal
//			depthBuffer[index].color = fragments[index].normal_eye_space;
//		}
//		else
//		{
//			depthBuffer[index].color = BACKGROUND_COLOR;
//		}
//	}
//}


// Writes fragment colors to the framebuffer
__host__ __device__
glm::vec3 phongShading(Light* lights, int num_lights
		, const glm::vec3 & pos, const glm::vec3 & n
		, const glm::vec3 & ambient, const glm::vec3 & diffuse,const glm::vec3 & specular, float shiniess
		)
{
	glm::vec3 ambient_term(0.0f);
	glm::vec3 diffuse_term(0.0f);
	glm::vec3 specular_term(0.0f);

	for (int i = 0; i < num_lights; i++)
	{
		if (!lights[i].enabled)
		{
			continue;
		}
		
		//ambient
		ambient_term += lights[i].intensity * ambient;

		//diffuse
		glm::vec3 l;
		if (lights[i].type == POINT_LIGHT)
		{
			l = glm::normalize(lights[i].vec - pos);
		}
		else
		{
			//directional light
			//should be normalized already
			l = -lights[i].vec;
		}
		diffuse_term += max(glm::dot(l, n), 0.0f) * diffuse * lights[i].intensity;
		

		//specular
		glm::vec3 v = glm::normalize(pos);	//?
		glm::vec3 r = (2 * glm::dot(l, n) * n) - l;
		//glm::vec3 h = (l + v) / glm::length(l + v);
		specular_term += powf(max(glm::dot(r,v), 0.0f), shiniess) * specular * lights[i].intensity;
	}

	return ambient_term + diffuse_term + specular_term;
}


__global__
void render(int w, int h, Fragment *depthbuffer, glm::vec3 *framebuffer,Light* lights,int num_lights, ShaderMode s) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		//framebuffer[index] = depthbuffer[index].color;
		Fragment & db = depthbuffer[index];

		if (!db.has_fragment)
		{
			framebuffer[index] = BACKGROUND_COLOR;
			return;
		}

		
		switch (s)
		{
		case SHADER_NORMAL:
		{
			framebuffer[index] = db.normal_eye_space;
			break;
		}
			
		case SHADER_WHITE_MATERIAL:
		{
			//using lights
			glm::vec3 zero(0.0f);
			framebuffer[index] = phongShading(lights,  num_lights
				, db.pos_eye_space, db.normal_eye_space
				, zero//glm::vec3(0.1f, 0.1f, 0.1f)
				, glm::vec3(1.0f, 0.0f, 0.0f)
				, glm::vec3(1.0f, 1.0f, 1.0f), 32
				);
			break;
		}

		case SHADER_TEXTURE:
		{
			//using texture
			//using lights
			break;
		}

		}
		
		
	}
}

//--------------------------------------------------------------------------------


__global__
void initDepth(int w,int h,int * depth)
{
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	
	if ( x < w && y < h)
	{
		int index = x + (y * w);

		depth[index] = INT_MAX;
	}

	
}


/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
    int sideLength2d = 8;
    dim3 blockSize2d(sideLength2d, sideLength2d);
    dim3 blockCount2d((width  - 1) / blockSize2d.x + 1,
                      (height - 1) / blockSize2d.y + 1);

    // TODO: Execute your rasterization pipeline here
    // (See README for rasterization pipeline outline.)

	//hipMemset(dev_fragments, 0, width * height * sizeof(FragmentIn));
	//hipMemset(dev_depth, INT_MAX, width * height * sizeof(int));
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));



	initDepth << <blockCount2d, blockSize2d >> >(width,height,dev_depth);


	//rasterization
	dim3 blockSize_Rasterize(64);
	dim3 blockCount_tri((triCount + blockSize_Rasterize.x - 1) / blockSize_Rasterize.x);

	hipDeviceSynchronize();
	kernScanLineForOneTriangle << <blockCount_tri, blockSize_Rasterize >> >(width, height, dev_primitives, dev_depthbuffer, dev_depth);


	//fragment shader
	//fragmentShader << <blockCount2d, blockSize2d >> >(width, height, dev_depthbuffer, dev_fragments);


    // Copy depthbuffer colors into framebuffer
	hipDeviceSynchronize();
    render<<<blockCount2d, blockSize2d>>>(width, height, dev_depthbuffer, dev_framebuffer,dev_lights, lightsCount, shaderMode);


    // Copy framebuffer into OpenGL buffer for OpenGL previewing
	hipDeviceSynchronize();
    sendImageToPBO<<<blockCount2d, blockSize2d>>>(pbo, width, height, dev_framebuffer);
	hipDeviceSynchronize();
    checkCUDAError("rasterize");
}



/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
    dev_bufIdx = NULL;

    hipFree(dev_bufVertex);
    dev_bufVertex = NULL;

    hipFree(dev_primitives);
    dev_primitives = NULL;

    hipFree(dev_depthbuffer);
    dev_depthbuffer = NULL;

	//hipFree(dev_fragments);
	//dev_fragments = NULL;

	hipFree(dev_depth);
	dev_depth = NULL;

    hipFree(dev_framebuffer);
    dev_framebuffer = NULL;


	hipFree(dev_lights);
	dev_lights = NULL;


    checkCUDAError("rasterizeFree");
}
