#include "hip/hip_runtime.h"
/**
 * @file      rasterize.cu
 * @brief     CUDA-accelerated rasterization pipeline.
 * @authors   Skeleton code: Yining Karl Li, Kai Ninomiya
 * @date      2012-2015
 * @copyright University of Pennsylvania & STUDENT
 */

#include <cmath>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stream_compaction/efficient.h>
#include "sceneStructs.h"
#include <util/checkCUDAError.h>
#include <glm/gtx/transform.hpp>
#include "rasterize.h"
#include "rasterizeTools.h"

#define BINSIDE_LEN 8
#define TILESIDE_LEN 8
#define BIN_SIZE BINSIDE_LEN*BINSIDE_LEN	// this many tiles
#define TILE_SIZE TILESIDE_LEN*TILESIDE_LEN	// this many pixels

#define BINRASTER_BLOCK 128
#define VERTSHADER_BLOCK 128
#define FRAGSHADER_BLOCK 256

#define QSEG_SIZE 1024

// Data structure for rasterization filter
namespace Queue {
	struct Segment {
		int queueSize = 0;
		int queue[QSEG_SIZE];
		Segment *next = NULL;
	};

	// LIMITATION: fixed length queue; need a real lockfree linked list
	__device__ void push(Segment &seg, int triId){
		int writeIdx = atomicAdd(&(seg.queueSize), 1);
		if (writeIdx < QSEG_SIZE){
			seg.queue[writeIdx] = triId;
		}
	}

	__device__ void clear(Segment &seg){
		atomicExch(&(seg.queueSize), 0);
	}
}

static int width = 0;
static int height = 0;
__constant__ static int *dev_bufIdx = NULL;
__constant__ static int *dev_depth = NULL;
__constant__ static VertexIn *dev_bufVertex = NULL;
__constant__ static VertexOut *dev_bufShadedVert = NULL;
__constant__ static Triangle *dev_primitives = NULL;
__constant__ static Fragment *dev_depthbuffer = NULL;
__constant__ static glm::vec3 *dev_framebuffer = NULL;

static int bufIdxSize = 0;
static int vertCount = 0;
static MVP *mvp = NULL;

// Geometry shader restriction
const int geomShaderLimit = 8;
static int triCount;

// Temp variables for stream compaction
__constant__ static int *dv_f_tmp = NULL;
__constant__ static int *dv_idx_tmp = NULL;
__constant__ static Triangle *dv_out_tmp = NULL;
__constant__ static int *dv_c_tmp = NULL;

// Fixed lighting
static glm::vec3 light1 = 10.0f*glm::vec3(100.0f, 100.0f, 100.0f);
static glm::vec3 lightCol1 = glm::vec3(0.95f, 0.95f, 1.0f);
static glm::vec3 light2 = light1 * glm::vec3(-1.0f, 1.0f, -1.0f);
static glm::vec3 lightCol2 = glm::vec3(1.0f, 0.725f, 0.494f);

// Rasterization filtering
int rowWidth;
int columnHeight;
int binGridWidth, binGridHeight;
__constant__ static Queue::Segment *binVsTriangle;
__constant__ static Queue::Segment *tileVsTriangle;

__global__ void sendImageToPBO(uchar4 *pbo, int w, int h, Fragment *image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * w);

	if (x < w && y < h) {
		Fragment f = image[index];
		glm::vec3 color = glm::vec3(255.0f);

		color.x = color.x * glm::clamp(f.col.x, 0.0f, 1.0f);
		color.y = color.y * glm::clamp(f.col.y, 0.0f, 1.0f);
		color.z = color.z * glm::clamp(f.col.z, 0.0f, 1.0f);
		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = color.x;
		pbo[index].y = color.y;
		pbo[index].z = color.z;
	}
}

/**
 * Called once at the beginning of the program to allocate memory.
 */
void rasterizeInit(int w, int h, MVP *hst_mvp) {
    width = w;
	height = h;
	mvp = hst_mvp;
    //hipFree(dev_depthbuffer);
	hipMalloc(&dev_depthbuffer, width * height * sizeof(Fragment));
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
    //hipFree(dev_framebuffer);
    hipMalloc(&dev_framebuffer,   width * height * sizeof(glm::vec3));
	hipMemset(dev_framebuffer, 0, width * height * sizeof(glm::vec3));

	hipMalloc(&dev_depth, width * height * sizeof(int));
	hipMemset(dev_depth, mvp->farPlane*10000, width * height * sizeof(int));
    checkCUDAError("rasterizeInit");
}

void flushDepthBuffer(){
	hipMemset(dev_depth, mvp->farPlane * 10000, width * height * sizeof(int));
	hipMemset(dev_depthbuffer, 0, width * height * sizeof(Fragment));
	hipMemset(dev_primitives, 0, triCount * geomShaderLimit * sizeof(Triangle));
	checkCUDAError("rasterize flush");
}

/**
 * Set all of the buffers necessary for rasterization.
 */
void rasterizeSetBuffers(
        int _bufIdxSize, int *bufIdx,
        int _vertCount, float *bufPos, float *bufNor, float *bufCol) {
    bufIdxSize = _bufIdxSize;
    vertCount = _vertCount;

    //hipFree(dev_bufIdx);
    hipMalloc(&dev_bufIdx, bufIdxSize * sizeof(int));
    hipMemcpy(dev_bufIdx, bufIdx, bufIdxSize * sizeof(int), hipMemcpyHostToDevice);

    VertexIn *bufVertex = new VertexIn[_vertCount];
    for (int i = 0; i < vertCount; i++) {
        int j = i * 3;
        bufVertex[i].pos = glm::vec3(bufPos[j + 0], bufPos[j + 1], bufPos[j + 2]);
        bufVertex[i].nor = glm::vec3(bufNor[j + 0], bufNor[j + 1], bufNor[j + 2]);
        bufVertex[i].col = glm::vec3(bufCol[j + 0], bufCol[j + 1], bufCol[j + 2]);
    }
    //hipFree(dev_bufVertex);
    hipMalloc(&dev_bufVertex, vertCount * sizeof(VertexIn));
    hipMemcpy(dev_bufVertex, bufVertex, vertCount * sizeof(VertexIn), hipMemcpyHostToDevice);

	//hipFree(dev_bufShadedVert);
	hipMalloc(&dev_bufShadedVert, vertCount * sizeof(VertexOut));

	triCount = vertCount / 3;

    //hipFree(dev_primitives);
    hipMalloc(&dev_primitives, triCount * geomShaderLimit * sizeof(Triangle));
	hipMemset(dev_primitives, 0, triCount * geomShaderLimit * sizeof(Triangle));

	// Allocate temp vars
	hipMalloc((void**)&dv_f_tmp, triCount * geomShaderLimit *sizeof(int));
	hipMalloc((void**)&dv_idx_tmp, triCount * geomShaderLimit *sizeof(int));
	hipMalloc((void**)&dv_out_tmp, triCount * geomShaderLimit *sizeof(Triangle));
	hipMalloc((void**)&dv_c_tmp, sizeof(int));

    checkCUDAError("rasterizeSetBuffers");
}

__global__ void shadeVertex(VertexOut *vOut, VertexIn *vIn, const int vertCount, const int width, const int height, const glm::mat4 mvp, const float near, const float far){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * width);

	if (index < vertCount) {
		// http://www.songho.ca/opengl/gl_transform.html
		VertexOut o;
		o.mpos = vIn[index].pos;
		o.nor = vIn[index].nor;
		o.col = vIn[index].col;
		glm::vec4 clip = mvp*glm::vec4(vIn[index].pos, 1.0f);
		glm::vec3 ndc = glm::vec3(clip.x / clip.w, clip.y / clip.w, clip.z / clip.w);
		o.pos = glm::vec3(
			width*0.5f*(ndc.x+1), 
			height*0.5f*(ndc.y+1), 
			((far-near)*ndc.z+(far+near))*0.5f
			);
		vOut[index] = o;
	}
}

__global__ void assemblePrimitive(Triangle *pOut, VertexOut *vIn, int *triIdx, const int triCount, const int width){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < triCount) {
		Triangle t;
		// Set rasterization property
		t.isPoint = false;
		t.isLine = false;
		t.isValidGeom = true;
		// Assemble vertices
		t.v[0] = vIn[triIdx[3 * index + 0]];
		t.v[1] = vIn[triIdx[3 * index + 1]];
		t.v[2] = vIn[triIdx[3 * index + 2]];
		// Find bounding box
		t.box = getAABBForTriangle(t);
		pOut[index] = t;
	}
}

__global__ void assemblePrimitivePoint(Triangle *pOut, VertexOut *vIn, int *triIdx, const int triCount, const int width){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < triCount) {
		Triangle t;
		t.v[0] = vIn[triIdx[3 * index + 0]];
		t.isPoint = true;
		t.isValidGeom = true;
		pOut[index] = t;
	}
}

__global__ void simpleShadeGeom(Triangle *pArr, const int triCount, const int width, const int limit, const int height, const glm::mat4 mvp, const float near, const float far){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * width);

	if (index < triCount) {
		Triangle t = pArr[index];
		Triangle tN = t;
		// Calculate a line that represents the vertex normal
		// Since normal is not MVP-transformed, need to do MVP here for the model-space normal line
		glm::vec4 clip = mvp*glm::vec4(t.v[0].mpos + t.v[0].nor*0.1f, 1.0f);
		glm::vec3 ndc = glm::vec3(clip.x / clip.w, clip.y / clip.w, clip.z / clip.w);
		tN.v[1].pos = glm::vec3(
			width / 2 * (ndc.x + 1),
			height / 2 * (ndc.y + 1),
			(far - near) / 2 * ndc.z + (far + near) / 2
			);
		tN.isLine = true;
		tN.isValidGeom = true;
		pArr[index + triCount] = tN;
	}
}

__global__ void simpleCulling(Triangle *pArr, const int triCount, const int width, const glm::vec3 camPos){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * width);

	if (index < triCount) {
		if (glm::dot(pArr[index].v[0].mpos - camPos, pArr[index].v[0].nor) >= 0){
			pArr[index].isValidGeom = false;
		}
	}
}

__global__ void testCover(Fragment *dBuf, int *depth, Triangle *pIn, const int triCount, const int width, const int height, const glm::vec3 camPos, const bool doScissor, const Scissor scissor, const glm::vec3 camLook){
	int xt = (blockIdx.x * blockDim.x) + threadIdx.x;
	int yt = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = xt + (yt * width);

	if (index < triCount) {
		if (pIn[index].isPoint){
			bool discard = false;
			int x = round(pIn[index].v[0].pos.x), y = round(pIn[index].v[0].pos.y);
			int flatIdx = width - x + (height - y)*width;
			// Scissor test
			if (doScissor){
				if (x > scissor.max.x || x < scissor.min.x || y > scissor.max.y || y < scissor.min.y){
					discard = true;
				}
			}
			// Window clipping test
			if (y < 0 || height < y || width < x || x < 0){
				discard = true;
			}
			if (!discard){
				int dp = -pIn[index].v[0].pos.z * 10000;
				// Try to win the depth test
				atomicMin(&depth[flatIdx], dp);
				// If won depth test
				if (depth[flatIdx] == dp) {
					// Shallowest
					Fragment f;
					f.col = pIn[index].v[0].col;
					f.nor = pIn[index].v[0].nor;
					f.pos = pIn[index].v[0].pos;
					dBuf[flatIdx] = f;
				}
			}
		}
		else if (pIn[index].isLine){
			glm::vec3 min = pIn[index].v[0].pos, max = pIn[index].v[1].pos;
			int minX = round(min.x), maxX = round(max.x);
			if (minX == maxX){
				// Straight vertical line
				int minY = round(min.y), maxY = round(max.y), minZ = min.z, maxZ = max.z;
				int x = minX;
				if (min.y > max.y){
					minY = round(max.y); maxY = round(min.y); minZ = max.z; maxZ = min.z;
				}
				int dp;
				bool discard;
				for (int y = maxY; y >= minY; y--){
					discard = false;
					// Scissor test
					if (doScissor){
						if (x > scissor.max.x || x < scissor.min.x || y > scissor.max.y || y < scissor.min.y){
							discard = true;
						}
					}
					int flatIdx = width - x + (height - y)*width;
					if (y < 0 || height < y || x > width || x < 0){
						discard = true;
					}
					if (!discard){
						float ratio = (y - minY) / (maxY - minY);
						dp = -(ratio*minZ + (1 - ratio)*maxZ) * 10000;

						atomicMin(&depth[flatIdx], dp);

						if (depth[flatIdx] == dp) {
							// Shallowest
							Fragment f;
							f.pos = glm::vec3(x, y, -dp* 0.0001);
							f.nor = glm::normalize(pIn[index].v[0].nor + pIn[index].v[1].nor);
							f.col = glm::vec3(1.0f);
							dBuf[flatIdx] = f;
						}
					}
				}
			}
			else {
				// Bresenham
				if (minX > maxX){
					min = pIn[index].v[1].pos; max = pIn[index].v[0].pos;
				}
				int minZ = min.z, maxZ = max.z;
				float slope = (max.y - min.y) / (max.x - min.x);
				int dp, y;
				bool discard;
				float ratio;
				for (int x = round(min.x); x <= round(max.x); x++){
					y = slope * (x - round(min.x)) + min.y;
					ratio = (x - round(min.x)) / (round(max.x) - round(min.x));
					discard = false;
					// Scissor test
					if (doScissor){
						if (x > scissor.max.x || x < scissor.min.x || y > scissor.max.y || y < scissor.min.y){
							discard = true;
						}
					}
					int flatIdx = width - x + (height - y)*width;
					if (y < 0 || y > height || x < 0 || x > width){
						discard = true;
					}
					if (!discard){
						dp = -(ratio*minZ + (1 - ratio)*maxZ) * 10000;

						atomicMin(&depth[flatIdx], dp);

						if (depth[flatIdx] == dp) {
							// Shallowest
							Fragment f;
							f.pos = glm::vec3(x, y, -dp*0.0001);
							f.nor = glm::normalize(pIn[index].v[0].nor + pIn[index].v[1].nor);
							f.col = glm::vec3(1.0f);
							dBuf[flatIdx] = f;
						}
					}
				}
			}
		}
		else {
			// General triangle
			// Early window clipping & scissor test
			int minX, maxX, minY, maxY;
			minX = fmaxf(round(pIn[index].box.min.x), 0.0f), maxX = fminf(round(pIn[index].box.max.x), (float)width);
			minY = fmaxf(round(pIn[index].box.min.y), 0.0f), maxY = fminf(round(pIn[index].box.max.y), (float)height);
			if (doScissor){
				minX = fmaxf(minX, scissor.min.x), maxX = fminf(maxX, scissor.max.x);
				minY = fmaxf(minY, scissor.min.y), maxY = fminf(maxY, scissor.max.y);
			}
			glm::vec3 coord[3] = { pIn[index].v[0].pos, pIn[index].v[1].pos, pIn[index].v[2].pos };
			int dp, flatIdx;
			glm::vec3 bcc;
			// For each scanline
			for (int y = maxY; y >= minY; y--){
				// Scan each pixel
				for (int x = minX; x <= maxX; x++){
					bcc = calculateBarycentricCoordinate(coord, glm::vec2(x, y));
					flatIdx = width - x + (height - y)*width;
					if (isBarycentricCoordInBounds(bcc)){
						dp = getZAtCoordinate(bcc, coord) * 10000;

						atomicMin(&depth[flatIdx], dp);

						if (depth[flatIdx] == dp) {
							// Shallowest
							Fragment f;
							f.pos = bcc.x * pIn[index].v[0].pos + bcc.y*pIn[index].v[1].pos + bcc.z*pIn[index].v[2].pos;
							f.nor = bcc.x * pIn[index].v[0].nor + bcc.y*pIn[index].v[1].nor + bcc.z*pIn[index].v[2].nor;
							f.col = bcc.x * pIn[index].v[0].col + bcc.y*pIn[index].v[1].col + bcc.z*pIn[index].v[2].col;
							dBuf[flatIdx] = f;
						}
					}
				}
			}
		}
	}
}

__global__ void shadeFragment(Fragment *fBuf, const int pxCount, const int width, const glm::vec3 light1, const glm::vec3 lightCol1, const glm::vec3 light2, const glm::vec3 lightCol2){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < pxCount) {
		// Add the two lights and do Lambert shading
		glm::vec3 L1 = glm::normalize(light1 - fBuf[index].pos);
		glm::vec3 L2 = glm::normalize(light2 - fBuf[index].pos);
		glm::vec3 C1 = glm::dot(L1, fBuf[index].nor)*fBuf[index].col*lightCol1;
		glm::vec3 C2 = glm::dot(L2, fBuf[index].nor)*fBuf[index].col*lightCol2;
		fBuf[index].col = C1+C2;
	}
}

__global__ void shadeFragmentNormal(Fragment *fBuf, const int pxCount, const int width){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < pxCount) {
		fBuf[index].col = fBuf[index].nor;
	}
}

/**
 * Perform rasterization.
 */
void rasterize(uchar4 *pbo) {
	int sideLength2d = 8;
	dim3 blockSize2d(sideLength2d, sideLength2d);

	dim3 blockCount2d((width + blockSize2d.x - 1) / blockSize2d.x,
		(height + blockSize2d.y - 1) / blockSize2d.y);

	int vertGridSize = (width*height + VERTSHADER_BLOCK - 1) / VERTSHADER_BLOCK;

	// Vertex shading
	shadeVertex << <vertGridSize, VERTSHADER_BLOCK>> >(dev_bufShadedVert, dev_bufVertex, vertCount, width, height, mvp->mvp, mvp->nearPlane, mvp->farPlane);
	checkCUDAError("Vert shader");

	// Primitive assembly
	if (mvp->pointShading){
		assemblePrimitivePoint << <vertGridSize, VERTSHADER_BLOCK >> >(dev_primitives, dev_bufShadedVert, dev_bufIdx, triCount, width);
		checkCUDAError("Prim assembly");
	}
	else {
		assemblePrimitive << <vertGridSize, VERTSHADER_BLOCK >> >(dev_primitives, dev_bufShadedVert, dev_bufIdx, triCount, width);
		checkCUDAError("Prim assembly");
	}
	
	int primCount = triCount;
	if (mvp->geomShading){
		simpleShadeGeom << <blockCount2d, blockSize2d >> >(dev_primitives, primCount, width, geomShaderLimit, height, mvp->mvp, mvp->nearPlane, mvp->farPlane);
		checkCUDAError("Geom shader");
		StreamCompaction::Efficient::compact(triCount*geomShaderLimit, dv_f_tmp, dv_idx_tmp, dv_out_tmp, dev_primitives, dv_c_tmp);
		checkCUDAError("Geom shader compact");
		hipMemcpy(&primCount, dv_c_tmp, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(dev_primitives, dv_out_tmp, primCount * sizeof(Triangle), hipMemcpyDeviceToDevice);
		checkCUDAError("Geom shader copy");
	}

	if (mvp->culling){
		simpleCulling << <blockCount2d, blockSize2d >> >(dev_primitives, primCount, width, mvp->camPosition);
		checkCUDAError("Culling");
		StreamCompaction::Efficient::compact(triCount*geomShaderLimit, dv_f_tmp, dv_idx_tmp, dv_out_tmp, dev_primitives, dv_c_tmp);
		checkCUDAError("Culling compact");
		hipMemcpy(&primCount, dv_c_tmp, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(dev_primitives, dv_out_tmp, primCount * sizeof(Triangle), hipMemcpyDeviceToDevice);
		checkCUDAError("Culling copy");
	}

	// Rasterization
	testCover << <blockCount2d, blockSize2d >> >(dev_depthbuffer, dev_depth, dev_primitives, primCount, width, height, mvp->camPosition, mvp->doScissor, mvp->scissor, mvp->camLookAt);
	checkCUDAError("Rasterization");

	int fragGridSize = (width*height + FRAGSHADER_BLOCK - 1) / FRAGSHADER_BLOCK;

	if (mvp->shadeMode == 0){
		// Fragment shading
		shadeFragment << <fragGridSize, FRAGSHADER_BLOCK >> >(dev_depthbuffer, height*width, width, light1, lightCol1, light2, lightCol2);
		checkCUDAError("Frag shader");
	}
	else if (mvp->shadeMode == 1){
		// Fragment shading
		shadeFragmentNormal << <fragGridSize, FRAGSHADER_BLOCK >> >(dev_depthbuffer, height*width, width);
		checkCUDAError("Frag shader");
	}

	dim3 blockSize2d2(16, 16);

	dim3 blockCount2d2((width + blockSize2d2.x - 1) / blockSize2d2.x,
		(height + blockSize2d2.y - 1) / blockSize2d2.y);

	sendImageToPBO << <blockCount2d2, blockSize2d2 >> >(pbo, width, height, dev_depthbuffer);
    checkCUDAError("rasterize");
}

/**
 * Called once at the end of the program to free CUDA memory.
 */
void rasterizeFree() {
    hipFree(dev_bufIdx);
	hipFree(dev_bufVertex);
	hipFree(dev_bufShadedVert);
    hipFree(dev_primitives);
    hipFree(dev_depthbuffer);
    hipFree(dev_framebuffer);
	hipFree(dev_depth);

	hipFree(dv_f_tmp);
	hipFree(dv_idx_tmp);
	hipFree(dv_out_tmp);
	hipFree(dv_c_tmp);

	hipFree(binVsTriangle);
	hipFree(tileVsTriangle);

    checkCUDAError("rasterizeFree");
}


/****************************************************************************************************************************************
****************************************************************************************************************************************
****************************************************************************************************************************************
****************************************************************************************************************************************
* Tile-based pipeline below
****************************************************************************************************************************************
****************************************************************************************************************************************
****************************************************************************************************************************************
*****************************************************************************************************************************************/

void rasterizeTileInit(){
	// Initialize tile arrays
	rowWidth = triCount * geomShaderLimit * sizeof(bool);
	binGridWidth = (width + BINSIDE_LEN*TILESIDE_LEN - 1) / (BINSIDE_LEN*TILESIDE_LEN);
	binGridHeight = (height + BINSIDE_LEN*TILESIDE_LEN - 1) / (BINSIDE_LEN*TILESIDE_LEN);
	hipMalloc((void**)&binVsTriangle, binGridHeight*binGridWidth*sizeof(Queue::Segment));
	checkCUDAError("Bin array");
	hipMalloc((void**)&tileVsTriangle, binGridHeight*binGridWidth*BIN_SIZE*sizeof(Queue::Segment));
	checkCUDAError("Tile array");
}

__global__ void assemblePrimitiveT(Triangle *pOut, VertexOut *vIn, int *triIdx, const int triCount, const int width, const int height, const glm::vec3 camPos, const bool doScissor, const Scissor scissor){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < triCount) {
		Triangle t;
		int base = 3 * index;
		// Set rasterization property
		t.isPoint = false; t.isLine = false; t.isValidGeom = true;
		// Assemble vertices
		t.v[0] = vIn[triIdx[base]];
		t.v[1] = vIn[triIdx[base + 1]];
		t.v[2] = vIn[triIdx[base + 2]];
		// Snapping
		// Revert coordinates to fix OpenGL coord quirks
		t.v[0].pos = glm::vec3(width - ceil(t.v[0].pos.x), height - ceil(t.v[0].pos.y), t.v[0].pos.z);
		t.v[1].pos = glm::vec3(width - ceil(t.v[1].pos.x), height - ceil(t.v[1].pos.y), t.v[1].pos.z);
		t.v[2].pos = glm::vec3(width - ceil(t.v[2].pos.x), height - ceil(t.v[2].pos.y), t.v[2].pos.z);
		// Find bounding box
		t.box = getAABBForTriangle(t);
		// Backface culling & degenerate (zero area)
		// Calculate signed area for later use also
		t.signedArea = calculateSignedArea(t.v);
		if (t.signedArea >= 0){
			t.isValidGeom = false;
		}
		// Coarse window & scissor clipping
		if (doScissor){
			if (t.box.min.x > scissor.max.x || t.box.max.x < scissor.min.x || t.box.min.y > scissor.max.y || t.box.max.y < scissor.min.y){
				t.isValidGeom = false;
			}
		}
		if (t.box.min.x > width || t.box.max.x < 0 || t.box.min.y > height || t.box.max.y < 0){
			t.isValidGeom = false;
		}
		// Minimum Z of all 3 vertices; for quick depth test
		t.minDepth = glm::min(glm::min(-t.v[0].pos.z, -t.v[1].pos.z), -t.v[2].pos.z);
		pOut[index] = t;
	}
}

__global__ void assemblePrimitivePointT(Triangle *pOut, VertexOut *vIn, int *triIdx, const int triCount, const int width, const int height){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < triCount) {
		Triangle t;
		t.v[0] = vIn[triIdx[3 * index + 0]];
		t.v[0].pos = glm::vec3(width - ceil(t.v[0].pos.x), height - ceil(t.v[0].pos.y), t.v[0].pos.z);
		t.isPoint = true;
		t.isValidGeom = true;
		pOut[index] = t;
	}
}

__global__ void simpleShadeGeomT(Triangle *pArr, const int triCount, const int width, const int limit, const int height, const glm::mat4 mvp, const float near, const float far){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * width);

	if (index < triCount && pArr[index].isValidGeom) {
		Triangle t = pArr[index];
		Triangle tN = t;
		// Calculate a line that represents the vertex normal
		// Since normal is not MVP-transformed, need to do MVP here for the model-space normal line
		glm::vec4 clip = mvp*glm::vec4(t.v[0].mpos + t.v[0].nor*0.1f, 1.0f);
		glm::vec3 ndc = glm::vec3(clip.x / clip.w, clip.y / clip.w, clip.z / clip.w);
		// Rounding
		tN.v[1].pos = glm::round(glm::vec3(
			width / 2 * (ndc.x + 1),
			height / 2 * (ndc.y + 1),
			(far - near) / 2 * ndc.z + (far + near) / 2
			));
		tN.isLine = true;
		tN.isValidGeom = true;
		pArr[index + triCount] = tN;
	}
}

__device__ void boxOverlapTest(bool &result, AABB a, AABB b){
	if (a.max.x < b.min.x) {
		result = false;
	}
	else if (a.min.x > b.max.x){
		result = false;
	}
	else if (a.max.y < b.min.y){
		result = false;
	}
	else if (a.min.y > b.max.y) {
		result = false;
	}
	else {
		result = true;
	}
}

__global__ void binCover(Queue::Segment* binVsTriangle, Triangle *dev_primitives, const int primCount, const int width, const int height){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < primCount){
		Triangle t = dev_primitives[index];
		for (int b = 0; b < width*height; b++){
			int binX = b % width, binY = (b - binX) / width;
			AABB bin;
			bin.min.x = binX*BINSIDE_LEN*TILESIDE_LEN, bin.max.x = bin.min.x + BINSIDE_LEN*TILESIDE_LEN;
			bin.min.y = binY*BINSIDE_LEN*TILESIDE_LEN, bin.max.y = bin.min.y + BINSIDE_LEN*TILESIDE_LEN;
			if (t.isPoint){
				if (
					t.v[0].pos.x <= bin.max.x && t.v[0].pos.x >= bin.min.x &&
					t.v[0].pos.y <= bin.max.y && t.v[0].pos.y >= bin.min.y
					){
					Queue::push(binVsTriangle[b], index);
				}
			}
			else if (t.isLine){
				if (
					((t.v[0].pos.x <= bin.max.x && t.v[0].pos.x >= bin.min.x) || (t.v[1].pos.x <= bin.max.x && t.v[1].pos.x >= bin.min.y)) &&
					((t.v[0].pos.y <= bin.max.y && t.v[0].pos.y >= bin.min.y) || (t.v[1].pos.y <= bin.max.y && t.v[1].pos.y >= bin.min.y))
					){
					Queue::push(binVsTriangle[b], index);
				}
			}
			else {
				bool overlap;
				boxOverlapTest(overlap, t.box, bin);
				if (overlap){
					Queue::push(binVsTriangle[b], index);
				}
			}
		}
	}
}

__global__ void testTrig(Fragment *fBuf, Triangle *prim, const int primC){
	for (int p = 0; p < primC; p++){
		for (int i = 0; i < 3; i++){
			int idx = prim[p].v[i].pos.x + prim[p].v[i].pos.y * 800;
			fBuf[idx].col = glm::vec3(1.0f, 0.0f, 0.0f);
		}
	}
}

__global__ void testBin(Fragment *fBuf, Queue::Segment *binFlag, const int binGridSize, const int width){
	for (int i = 0; i < binGridSize; i++){
		if (binFlag[i].queueSize > 0){
			glm::vec3 col = glm::vec3(0.0f, 1.0f, 0.0f);
			if (binFlag[i].queueSize > QSEG_SIZE){
				col = glm::vec3(0.0f, 0.0f, 1.0f);
			}
			int binX = i % width, binY = (i - binX) / width;
			int minX = binX*BINSIDE_LEN*TILESIDE_LEN, maxX = minX + BINSIDE_LEN*TILESIDE_LEN;
			int minY = binY*BINSIDE_LEN*TILESIDE_LEN, maxY = minY + BINSIDE_LEN*TILESIDE_LEN;
			int idx = minX + minY * 800;
			for (int x = idx; x < minX + 4 + minY * 800; x++){
				fBuf[x].col = col;
			}
			for (int y = idx; y < minX + (minY + 4) * 800; y += 800){
				fBuf[y].col = col;
			}
		}
	}
}

__global__ void testTile(Fragment *fBuf, Queue::Segment *binFlag, const int binGridSize, const int width, const int max){
	for (int i = 0; i < binGridSize; i++){
		if (binFlag[i].queueSize > 0){
			glm::vec3 col = glm::vec3(1.0f);
			if (binFlag[i].queueSize > QSEG_SIZE){
				col = glm::vec3(1.0f, 1.0f, 0.0f);
			}
			int binX = i % width, binY = (i - binX) / width;
			int minX = binX*TILESIDE_LEN, maxX = minX + TILESIDE_LEN;
			int minY = binY*TILESIDE_LEN, maxY = minY + TILESIDE_LEN;
			if (minX < 800 && minY < 800){
				int idx = minX + minY * 800;
				if (idx < max){
					fBuf[idx].col = col;
					fBuf[idx + 1].col = col;
					fBuf[idx + 2].col = col;
					fBuf[idx + 800].col = col;
					fBuf[idx + 1600].col = col;
				}
			}
		}
	}
}

__global__ void testPx(Fragment *fBuf, Queue::Segment *tileFlag, Triangle *prim, const int binGridWidth, const int width, const int height){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * width);
	int tileIdx = x / TILESIDE_LEN + (y / TILESIDE_LEN)*binGridWidth*BINSIDE_LEN;

	if (x < width && y < height){
		Fragment f;
		if (tileFlag[tileIdx].queueSize > 0){
			f.isCovered = true;
			f.col = glm::vec3(1.0f);
			f.nor = glm::vec3(1.0f);
			f.pos = glm::vec3(x, y, 0);
		}
		else {
			f.isCovered = false;
			f.col = glm::vec3(0.0f);
			f.nor = glm::vec3(0.0f);
			f.pos = glm::vec3(x, y, 0);
		}
		tileFlag[tileIdx].queueSize = 0;
		fBuf[index] = f;
	}
}

__global__ void tileCover(Queue::Segment *tileVsTriangle, Queue::Segment *binVsTriangle, Triangle *dev_primitives, const int width){
	int binId = blockIdx.x;
	int binX = binId % width, binY = (binId - binX) / width;
	int baseTileX = binX * BINSIDE_LEN, baseTileY = binY * BINSIDE_LEN;
	int tileId = baseTileX + threadIdx.x + (baseTileY + threadIdx.y)*width*BINSIDE_LEN;
	int tileMinX = (baseTileX + threadIdx.x)*TILESIDE_LEN, tileMaxX = tileMinX + TILESIDE_LEN;
	int tileMinY = (baseTileY + threadIdx.y)*TILESIDE_LEN, tileMaxY = tileMinY + TILESIDE_LEN;

	AABB tile;
	tile.min.x = tileMinX; tile.max.x = tileMaxX; tile.min.y = tileMinY; tile.max.y = tileMaxY;

	int bound = binVsTriangle[binId].queueSize > QSEG_SIZE ? QSEG_SIZE : binVsTriangle[binId].queueSize;

	for (int i = 0; i < bound; i++){
		Triangle t = dev_primitives[binVsTriangle[binId].queue[i]];
		if (t.isPoint){
			if (
				t.v[0].pos.x <= tileMaxX && t.v[0].pos.x >= tileMinX &&
				t.v[0].pos.y <= tileMaxY && t.v[0].pos.y >= tileMinY
				){
				Queue::push(tileVsTriangle[tileId], binVsTriangle[binId].queue[i]);
			}
		}
		else if (t.isLine){
			if (
				((t.v[0].pos.x <= tileMaxX && t.v[0].pos.x >= tileMinX) || (t.v[1].pos.x <= tileMaxX && t.v[1].pos.x >= tileMinX)) &&
				((t.v[0].pos.y <= tileMaxY && t.v[0].pos.y >= tileMinY) || (t.v[1].pos.y <= tileMaxY && t.v[1].pos.y >= tileMinY))
				){
				Queue::push(tileVsTriangle[tileId], binVsTriangle[binId].queue[i]);
			}
		}
		else {
			bool overlap;
			boxOverlapTest(overlap, t.box, tile);
			if (overlap){
				Queue::push(tileVsTriangle[tileId], binVsTriangle[binId].queue[i]);
			}
		}
	}

	__syncthreads();
	if (threadIdx.x + threadIdx.y*BINSIDE_LEN == 0){
		Queue::clear(binVsTriangle[binId]);
	}
}

__global__ void pixCover(Fragment *dev_depthbuffer, Queue::Segment *tileVsTriangle, Triangle *dev_primitives, const int width, const int height, const int binGridWidth, const bool doScissor, const Scissor scissor){
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * width);
	int tileIdx = x/TILESIDE_LEN + (y/TILESIDE_LEN)*binGridWidth*BINSIDE_LEN;

	if (x < width && y < height) {
		bool discard = false;
		bool covered = false;
		Fragment f;
		if (doScissor){
			if (x < scissor.min.x || x > scissor.max.x || y < scissor.min.y || y > scissor.max.y){
				discard = true;
			}
		}
		if (!discard){
			float depth = 100;
			if (tileVsTriangle[tileIdx].queueSize > 0){
				covered = true;
			}
			for (int i = 0; i < tileVsTriangle[tileIdx].queueSize; i++){
				Triangle t = dev_primitives[tileVsTriangle[tileIdx].queue[i]];
				if (t.isPoint){
					if (t.v[0].pos.x == x && t.v[0].pos.y == y){
						if (-t.v[0].pos.z <= depth) {
							// Shallowest
							f.col = t.v[0].col;
							f.nor = t.v[0].nor;
							f.pos = t.v[0].pos;
							depth = -t.v[0].pos.z;
						}
					}
				}
				else if (t.isLine){
					glm::vec3 min = t.v[0].pos, max = t.v[1].pos;
					int minX = round(min.x), maxX = round(max.x);
					if (minX == maxX){
						// Straight vertical line
						if (x == minX){
							int minY = round(min.y), maxY = round(max.y), minZ = min.z, maxZ = max.z;
							if (min.y > max.y){
								minY = round(max.y); maxY = round(min.y); minZ = max.z; maxZ = min.z;
							}
							float ratio = (y - minY) / (maxY - minY);
							float dp = -(ratio*minZ + (1 - ratio)*maxZ);

							if (dp <= depth) {
								// Shallowest
								f.pos = glm::vec3(x, y, -dp* 0.0001);
								f.nor = glm::normalize(t.v[0].nor + t.v[1].nor);
								f.col = glm::vec3(1.0f);
								depth = dp;
							}
						}
					}
					else {
						// Bresenham
						if (minX > maxX){
							min = t.v[1].pos; max = t.v[0].pos;
						}
						int minZ = min.z, maxZ = max.z;
						float slope = (max.y - min.y) / (max.x - min.x);
						float ratio;
						int assumedY = slope * (x - round(min.x)) + min.y;
						if (assumedY == y){
							ratio = (x - round(min.x)) / (round(max.x) - round(min.x));
							float dp = -(ratio*minZ + (1 - ratio)*maxZ);

							if (dp <= depth) {
								// Shallowest
								f.pos = glm::vec3(x, y, -dp*0.0001);
								f.nor = glm::normalize(t.v[0].nor + t.v[1].nor);
								f.col = glm::vec3(1.0f);
								depth = dp;
							}
						}
					}
				}
				else {
					// General triangle
					glm::vec3 bcc = calculateBarycentricCoordinate(t, glm::vec2(x, y));
					if (isBarycentricCoordInBounds(bcc)){
						if (t.minDepth <= depth){
							float dp = getZAtCoordinate(bcc, t);
							if (dp <= depth) {
								// Shallowest
								f.pos = bcc.x * t.v[0].pos + bcc.y*t.v[1].pos + bcc.z*t.v[2].pos;
								f.nor = bcc.x * t.v[0].nor + bcc.y*t.v[1].nor + bcc.z*t.v[2].nor;
								f.col = bcc.x * t.v[0].col + bcc.y*t.v[1].col + bcc.z*t.v[2].col;
								depth = dp;
							}
						}
					}
				}
			}
		}
		f.isCovered = covered;
		dev_depthbuffer[index] = f;
	}
	__syncthreads();
	Queue::clear(tileVsTriangle[tileIdx]);
}

__global__ void shadeFragmentT(Fragment *fBuf, const int pxCount, const int width, const glm::vec3 light1, const glm::vec3 lightCol1, const glm::vec3 light2, const glm::vec3 lightCol2){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < pxCount) {
		if (fBuf[index].isCovered){
			// Add the two lights and do Lambert shading
			glm::vec3 L1 = glm::normalize(light1 - fBuf[index].pos);
			glm::vec3 L2 = glm::normalize(light2 - fBuf[index].pos);
			glm::vec3 C1 = glm::dot(L1, fBuf[index].nor)*fBuf[index].col*lightCol1;
			glm::vec3 C2 = glm::dot(L2, fBuf[index].nor)*fBuf[index].col*lightCol2;
			fBuf[index].col = C1 + C2;
		}
		else {
			fBuf[index].col = glm::vec3(0.0f);
		}
	}
}

__global__ void shadeFragmentNormalT(Fragment *fBuf, const int pxCount, const int width){
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if (index < pxCount) {
		fBuf[index].col = fBuf[index].nor;
	}
}

/**
* Perform rasterization.
*/
void rasterizeTile(uchar4 *pbo) {
	int sideLength2d = 8;
	dim3 blockSize2d(sideLength2d, sideLength2d);

	dim3 blockCount2d((width + blockSize2d.x - 1) / blockSize2d.x,
		(height + blockSize2d.y - 1) / blockSize2d.y);

	int vertGridSize = (width*height + VERTSHADER_BLOCK - 1) / VERTSHADER_BLOCK;

	// Vertex shading
	shadeVertex << <vertGridSize, VERTSHADER_BLOCK >> >(dev_bufShadedVert, dev_bufVertex, vertCount, width, height, mvp->mvp, mvp->nearPlane, mvp->farPlane);
	checkCUDAError("Vert shader");

	// Primitive assembly
	if (mvp->pointShading){
		assemblePrimitivePointT << <vertGridSize, VERTSHADER_BLOCK >> >(dev_primitives, dev_bufShadedVert, dev_bufIdx, triCount, width, height);
		checkCUDAError("Prim assembly");
	}
	else {
		assemblePrimitiveT << <vertGridSize, VERTSHADER_BLOCK >> >(dev_primitives, dev_bufShadedVert, dev_bufIdx, triCount, width, height, mvp->camPosition, mvp->doScissor, mvp->scissor);
		checkCUDAError("Prim assembly");
	}

	int primCount = triCount;

	// Primitive compaction
	StreamCompaction::Efficient::compact(triCount*geomShaderLimit, dv_f_tmp, dv_idx_tmp, dv_out_tmp, dev_primitives, dv_c_tmp);
	checkCUDAError("Primitive compact");
	hipMemcpy(&primCount, dv_c_tmp, sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(dev_primitives, dv_out_tmp, primCount * sizeof(Triangle), hipMemcpyDeviceToDevice);
	checkCUDAError("Primitive copy");

	// Geometry shading
	if (mvp->geomShading){
		simpleShadeGeomT << <blockCount2d, blockSize2d >> >(dev_primitives, primCount, width, geomShaderLimit, height, mvp->mvp, mvp->nearPlane, mvp->farPlane);
		checkCUDAError("Geom shader");
		StreamCompaction::Efficient::compact(triCount*geomShaderLimit, dv_f_tmp, dv_idx_tmp, dv_out_tmp, dev_primitives, dv_c_tmp);
		checkCUDAError("Geom shader compact");
		hipMemcpy(&primCount, dv_c_tmp, sizeof(int), hipMemcpyDeviceToHost);
		hipMemcpy(dev_primitives, dv_out_tmp, primCount * sizeof(Triangle), hipMemcpyDeviceToDevice);
		checkCUDAError("Geom shader copy");
	}

	// Rasterization
	// Input to bin raster
	int binCoverGridSize = (primCount + BINRASTER_BLOCK - 1) / BINRASTER_BLOCK;
	binCover << <binCoverGridSize, BINRASTER_BLOCK>> >(binVsTriangle, dev_primitives, primCount, binGridWidth, binGridHeight);
	checkCUDAError("Bin cover test");

	//testTrig << <1, 1 >> >(dev_depthbuffer, dev_primitives, primCount);
	//testBin << <1, 1 >> >(dev_depthbuffer, binVsTriangle, binGridHeight*binGridWidth, binGridWidth);

	// Bin to tile raster
	dim3 binSize2d(BINSIDE_LEN, BINSIDE_LEN);
	tileCover << <binGridHeight*binGridWidth, binSize2d >> >(tileVsTriangle, binVsTriangle, dev_primitives, binGridWidth);
	checkCUDAError("Tile cover test");

	//testTile << <1, 1 >> >(dev_depthbuffer, tileVsTriangle, binGridHeight*binGridWidth*BIN_SIZE, binGridWidth*BINSIDE_LEN, width*height);

	// Tile to fragment raster
	pixCover << <blockCount2d, blockSize2d >> >(dev_depthbuffer, tileVsTriangle, dev_primitives, width, height, binGridWidth, mvp->doScissor, mvp->scissor);
	checkCUDAError("Pixel cover test");

	//testPx << <blockCount2d, blockSize2d >> >(dev_depthbuffer, tileVsTriangle, dev_primitives, binGridWidth, width, height);

	// Fragment shading
	int fragGridSize = (width*height + FRAGSHADER_BLOCK - 1) / FRAGSHADER_BLOCK;

	if (mvp->shadeMode == 0){
		shadeFragmentT << <fragGridSize, FRAGSHADER_BLOCK >> >(dev_depthbuffer, height*width, width, light1, lightCol1, light2, lightCol2);
		checkCUDAError("Frag shader");
	}
	else if (mvp->shadeMode == 1){
		shadeFragmentNormalT << <fragGridSize, FRAGSHADER_BLOCK >> >(dev_depthbuffer, height*width, width);
		checkCUDAError("Frag shader");
	}

	// Render to frame
	dim3 blockSize2d2(16, 16);

	dim3 blockCount2d2((width + blockSize2d2.x - 1) / blockSize2d2.x,
		(height + blockSize2d2.y - 1) / blockSize2d2.y);

	sendImageToPBO << <blockCount2d2, blockSize2d2 >> >(pbo, width, height, dev_depthbuffer);
	checkCUDAError("rasterize");
}